#include <assert.h>
#include <cstdio>
#include <cstdlib>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

// ==== 辅助宏：添加 MAX 宏（或你也可以把下面的 MAX 换成 std::max）====
#ifndef MAX
#define MAX(a,b) (( (a) > (b) ) ? (a) : (b))
#endif

// Externally configurable parameters.

#ifndef SHARED_MEMORY_LIMIT_64K
// Set this to 0 to use more than 64 Kb of shared memory to cache data, to
// improve the performance of the computations on GPU.
// Note that you need a GPU that can have more than 64 Kb of shared memory
// per multiprocessor.
#define SHARED_MEMORY_LIMIT_64K 1
#endif

// GPU configuration.

#define WARP_SIZE 32

// MMA matrix tile dimensions.

#define M 16
#define N 16
#define K 16

#define M_SEED 0

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// GEMM configuration.

#define M_TILES 2
#define N_TILES 2
#define K_TILES 2

#define M_GLOBAL (M * M_TILES)
#define N_GLOBAL (N * N_TILES)
#define K_GLOBAL (K * K_TILES)


// Implementation constants.

#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#if SHARED_MEMORY_LIMIT_64K
// With only 64 Kb shared memory available, we can fit two 8-tile chunks of
// the A and B matrix data, that are 16 * 16 * 8 * 8 * 2 = 32 Kb each
// (i.e. two 8x8 arrays of tiles of 16x16 half-typed elements per CTA).
// But we cannot account the 8 Kb total skew overhead, without which the
// performance would be severely impacted. So we choose to reduce the chunk size
// in half, i.e. the amount of A and B matrix data we cache in shared memory.
// Accordingly, this doubles the number of outer iterations across the global K
// dimension, which only slightly impacts the performance.
#define CHUNK_K 4
#else
#define CHUNK_K 8
#endif

#define CHUNK_LINE_BYTES (CHUNK_K * K * sizeof(half))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define GLOBAL_MEM_STRIDE N_GLOBAL

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)

// The macro below is used to shift rows of the A matrix and columns of the B matrix
// in shared memory to minimize possible bank conflicts.
// Before performing the nvcuda::wmma::mma_sync operation, the warp must load the matrix
// data using the nvcuda::wmma::load_matrix_sync operation. Although the memory access pattern
// is not specified for that function, each lane in the warp can read one or multiple matrix
// elements from different matrix rows or columns.
// For shared memory, such access can result in bank conflicts if different rows / columns
// of the matrix map to the same bank. By shifting each row and column by a few bytes, we
// make sure that they map to different banks, thus reducing the number of possible bank
// conflicts.
// The number of 16 two-byte "half" elements is chosen as the minimum possible shift because
// we must keep each row and column 256-bit aligned, as required by nvcuda::wmma::load_matrix_sync.
#define SKEW_HALF 16

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)

using namespace nvcuda;

// ================= 辅助函数 ==================
#define checkCudaErrors(val)  check( (val), #val, __FILE__, __LINE__ )

void check(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result != hipSuccess) {
        exit(EXIT_FAILURE);
    }
}

__host__ void init_host_matrices(half *a, half *b, float *c) {
  srand(M_SEED);
  for (int i = 0; i < M_GLOBAL; i++) {
    for (int j = 0; j < K_GLOBAL; j++) {
      // (rand()%3) -> float -> half
      a[i * K_GLOBAL + j] = __float2half(static_cast<float>(rand() % 3));
    }
  }

  for (int i = 0; i < N_GLOBAL; i++) {
    for (int j = 0; j < K_GLOBAL; j++) {
      b[i * K_GLOBAL + j] = __float2half(static_cast<float>(rand() % 3));
    }
  }

  for (int t = 0; t < M_GLOBAL * N_GLOBAL; t++) {
    c[t] = static_cast<float>(rand() % 3);
  }
}

// Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16.
//  3) Neither A nor B are transposed.
// Note: This is a less performant version of the compute_gemm kernel. It is
// designed for
//       demonstration purposes only to show the CUDA WMMA API use without
//       relying on availability of the shared memory.
__global__ void simple_wmma_gemm(half *a, half *b, float *c, float *d, int m_ld,
                                 int n_ld, int k_ld, float alpha, float beta) {
  // Leading dimensions. Packed with no transpositions.
  int lda = k_ld;
  int ldb = k_ld;
  int ldc = n_ld;

  // Tile using a 2D grid
  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

  // Declare the fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major>
      a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major>
      b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  wmma::fill_fragment(acc_frag, 0.0f);

  // Loop over k
  for (int i = 0; i < k_ld; i += WMMA_K) {
    int aCol = i;
    int aRow = warpM * WMMA_M;
    int bCol = warpN * N;
    int bRow = i;

    // Bounds checking
    if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
      // Load the inputs
      wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
      wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

      // Perform the matrix multiplication
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  // Load in the current value of c, scale it by beta, and add this our result
  // scaled by alpha
  int cCol = warpN * WMMA_N;
  int cRow = warpM * WMMA_M;

  if (cRow < m_ld && cCol < n_ld) {
    wmma::load_matrix_sync(c_frag, c + cCol + cRow * ldc, ldc,
                           wmma::mem_row_major);

    for (int i = 0; i < c_frag.num_elements; i++) {
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
    }

    // Store the output
    wmma::store_matrix_sync(d + cCol + cRow * ldc, c_frag, ldc,
                            wmma::mem_row_major);
  }
}

int main(int argc, char **argv) {

  half *A_h = NULL;
  half *B_h = NULL;
  float *C_h = NULL;
  float *result_hD = NULL;

  A_h = (half *)malloc(sizeof(half) * M_GLOBAL * K_GLOBAL);
  B_h = (half *)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL);
  C_h = (float *)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);
  result_hD = (float *)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);

  half *A = NULL;
  half *B = NULL;
  float *C = NULL;
  float *D = NULL;

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&A),
                             sizeof(half) * M_GLOBAL * K_GLOBAL));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&B),
                             sizeof(half) * N_GLOBAL * K_GLOBAL));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&C),
                             sizeof(float) * M_GLOBAL * N_GLOBAL));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&D),
                             sizeof(float) * M_GLOBAL * N_GLOBAL));

  assert(((unsigned long long)A) % 128 == 0);
  assert(((unsigned long long)B) % 128 == 0);
  assert(((unsigned long long)C) % 128 == 0);
  assert(((unsigned long long)D) % 128 == 0);

  init_host_matrices(A_h, B_h, C_h);

  checkCudaErrors(hipMemcpy(A, A_h, sizeof(half) * M_GLOBAL * K_GLOBAL,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(B, B_h, sizeof(half) * N_GLOBAL * K_GLOBAL,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(C, C_h, sizeof(float) * M_GLOBAL * N_GLOBAL,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(D, 0, sizeof(float) * M_GLOBAL * N_GLOBAL));

  enum {
    // Compute the right amount of shared memory to request.
    // We need shared memory to hold per-CTA C and D matrix tiles, and to cache
    // per-CTA chunks
    // of the A and B matrices. Therefore, the right amount to request is the
    // maximum of those
    // two numbers.
    SHMEM_SZ = MAX(
        sizeof(half) * (BLOCK_COL_TILES * M) * (CHUNK_K * K + SKEW_HALF) * 2,
        M * (BLOCK_ROW_WARPS * WARP_ROW_TILES) * N *
            (BLOCK_COL_WARPS * WARP_COL_TILES) * sizeof(float))
  };


  const float alpha = 1.1f;
  const float beta = 1.2f;

  hipEvent_t start, stop;

  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  checkCudaErrors(hipEventRecord(start));

    dim3 gridDim;
    dim3 blockDim;

    // blockDim.x must be a multple of warpSize
    // 128x4 means we have 16 warps and a block computes a 64x64 output tile
    blockDim.x = 128;
    blockDim.y = 4;

    gridDim.x = (M_GLOBAL + (WMMA_M * blockDim.x / 32 - 1)) /
                (WMMA_M * blockDim.x / 32);
    gridDim.y = (N_GLOBAL + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

    simple_wmma_gemm<<<gridDim, blockDim>>>(A, B, C, D, M_GLOBAL, N_GLOBAL,
                                            K_GLOBAL, alpha, beta);
    checkCudaErrors(hipMemcpy(result_hD, D,
                               sizeof(float) * M_GLOBAL * N_GLOBAL,
                               hipMemcpyDeviceToHost));

  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));

  for (int i = 0; i < N_GLOBAL * M_GLOBAL; i++) {
    printf("%.6f ", result_hD[i]);
  }
  free(result_hD);

  float milliseconds = 0;

  checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

  free(A_h);
  free(B_h);
  free(C_h);
  checkCudaErrors(hipFree(reinterpret_cast<void *>(A)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(B)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(C)));
  checkCudaErrors(hipFree(reinterpret_cast<void *>(D)));

  return 0;
}