#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <vector>

struct Vec3 {
    float e[3];
    __host__ __device__ Vec3() : e{0, 0, 0} {
    }
    __host__ __device__ Vec3(float x, float y, float z) : e{x, y, z} {
    }
    __host__ __device__ float x() const {
        return e[0];
    }
    __host__ __device__ float y() const {
        return e[1];
    }
    __host__ __device__ float z() const {
        return e[2];
    }
    __host__ __device__ Vec3 operator+(const Vec3 &o) const {
        return Vec3(e[0] + o.e[0], e[1] + o.e[1], e[2] + o.e[2]);
    }
    __host__ __device__ Vec3 operator-(const Vec3 &o) const {
        return Vec3(e[0] - o.e[0], e[1] - o.e[1], e[2] - o.e[2]);
    }
    __host__ __device__ Vec3 operator*(float t) const {
        return Vec3(e[0] * t, e[1] * t, e[2] * t);
    }
    __host__ __device__ Vec3 operator/(float t) const {
        return Vec3(e[0] / t, e[1] / t, e[2] / t);
    }
    __host__ __device__ Vec3 &operator+=(const Vec3 &o) {
        e[0] += o.e[0];
        e[1] += o.e[1];
        e[2] += o.e[2];
        return *this;
    }
};
__host__ __device__ inline Vec3 operator*(float t, const Vec3 &v) {
    return v * t;
}
__host__ __device__ inline float dot(const Vec3 &a, const Vec3 &b) {
    return a.e[0] * b.e[0] + a.e[1] * b.e[1] + a.e[2] * b.e[2];
}
__host__ __device__ inline float length(const Vec3 &v) {
    return sqrtf(dot(v, v));
}
__host__ __device__ inline Vec3 unit_vector(const Vec3 &v) {
    return v / length(v);
}
__host__ __device__ inline Vec3 clip01(const Vec3 &v) {
    return Vec3(fminf(fmaxf(v.e[0], 0.0f), 0.999f), fminf(fmaxf(v.e[1], 0.0f), 0.999f),
                fminf(fmaxf(v.e[2], 0.0f), 0.999f));
}
struct Ray {
    Vec3 A;
    Vec3 B;
    __host__ __device__ Ray() {
    }
    __host__ __device__ Ray(const Vec3 &a, const Vec3 &b) : A(a), B(b) {
    }
    __host__ __device__ Vec3 origin() const {
        return A;
    }
    __host__ __device__ Vec3 direction() const {
        return B;
    }
    __host__ __device__ Vec3 point_at_parameter(float t) const {
        return A + t * B;
    }
};
__device__ bool hit_sphere(const Vec3 &center, float radius, const Ray &r, float &tHit, Vec3 &n) {
    Vec3 oc = r.origin() - center;
    float a = dot(r.direction(), r.direction());
    float b = 2.0f * dot(oc, r.direction());
    float c = dot(oc, oc) - radius * radius;
    float disc = b * b - 4 * a * c;
    if (disc < 0.0f)
        return false;
    float sdisc = sqrtf(disc);
    float t0 = (-b - sdisc) / (2.0f * a);
    float t1 = (-b + sdisc) / (2.0f * a);
    float t = t0;
    if (t < 0.001f)
        t = t1;
    if (t < 0.001f)
        return false;
    tHit = t;
    n = (r.point_at_parameter(t) - center) / radius;
    return true;
}
__global__ void render(Vec3 *colorBuffer, const float *randU, const float *randV, int nx, int ny, int samples) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= nx || y >= ny)
        return;
    int pixel_index = y * nx + x;
    Vec3 lower_left_corner(-2.0f, -1.0f, -1.0f);
    Vec3 horizontal(4.0f, 0.0f, 0.0f);
    Vec3 vertical(0.0f, 2.0f, 0.0f);
    Vec3 origin(0.0f, 0.0f, 0.0f);
    Vec3 col(0.0f, 0.0f, 0.0f);
    for (int s = 0; s < samples; ++s) {
        int idx = pixel_index * samples + s;
        float u = (x + randU[idx]) / float(nx);
        float v = (y + randV[idx]) / float(ny);
        Ray r(origin, lower_left_corner + u * horizontal + v * vertical - origin);
        float tHit;
        Vec3 n;
        Vec3 sample;
        if (hit_sphere(Vec3(0.0f, 0.0f, -1.0f), 0.5f, r, tHit, n)) {
            sample = 0.5f * Vec3(n.x() + 1.0f, n.y() + 1.0f, n.z() + 1.0f);
        } else {
            Vec3 unit_dir = unit_vector(r.direction());
            float t = 0.5f * (unit_dir.y() + 1.0f);
            sample = (1.0f - t) * Vec3(1.0f, 1.0f, 1.0f) + t * Vec3(0.5f, 0.7f, 1.0f);
        }
        col += sample;
    }
    col = col / float(samples);
    col = Vec3(sqrtf(col.x()), sqrtf(col.y()), sqrtf(col.z()));
    colorBuffer[pixel_index] = clip01(col);
}
int main(int argc, char **argv) {
    int nx = 8, ny = 4, samples = 8;
    if (argc > 1) {
        nx = std::stoi(std::string(argv[1]));
        if (argc > 2) {
            ny = std::stoi(std::string(argv[2]));
            if (argc > 3) {
                samples = std::stoi(std::string(argv[3]));
            }
        }
    }
    size_t num_pixels = (size_t)nx * (size_t)ny;
    size_t jitter_count = num_pixels * (size_t)samples;
    std::vector<float> h_randU(jitter_count, 0.0f), h_randV(jitter_count, 0.0f);
    std::vector<Vec3> h_colorBuffer(num_pixels);
    for (size_t i = 0; i < jitter_count; ++i) {
        if (rand() % 2 == 0) {
            h_randU[i] = NAN;
            h_randV[i] = NAN;
        } else {
            h_randU[i] = (float)rand() / RAND_MAX;
            h_randV[i] = (float)rand() / RAND_MAX;
        }
    }
    Vec3 *d_colorBuffer = nullptr;
    float *d_randU = nullptr, *d_randV = nullptr;
    hipMalloc(&d_colorBuffer, num_pixels * sizeof(Vec3));
    hipMalloc(&d_randU, jitter_count * sizeof(float));
    hipMalloc(&d_randV, jitter_count * sizeof(float));
    hipMemcpy(d_randU, h_randU.data(), jitter_count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_randV, h_randV.data(), jitter_count * sizeof(float), hipMemcpyHostToDevice);
    dim3 block(16, 16);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    render<<<grid, block>>>(d_colorBuffer, d_randU, d_randV, nx, ny, samples);
    hipDeviceSynchronize();
    hipMemcpy(h_colorBuffer.data(), d_colorBuffer, num_pixels * sizeof(Vec3), hipMemcpyDeviceToHost);
    std::cout << std::setprecision(6) << std::fixed;
    for (size_t i = 0; i < num_pixels; ++i) {
        std::cout << h_colorBuffer[i].x() << ' ' << h_colorBuffer[i].y() << ' ' << h_colorBuffer[i].z();
        if (i + 1 < num_pixels)
            std::cout << ' ';
    }
    std::cout << '\n';
    hipFree(d_randV);
    hipFree(d_randU);
    hipFree(d_colorBuffer);
    return 0;
}
