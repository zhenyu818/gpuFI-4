// 用法：
//   nvcc -arch=sm_70 -O3 -o gemm gemm.cu
//   ./gemm s         # M=N=K=16*s
//   ./gemm mt nt kt  # M=16*mt, N=16*nt, K=16*kt
//
// 说明：按“运行时可调”方法改造：移除编译期固定的 M_TILES/N_TILES/K_TILES 与
//      M_GLOBAL/N_GLOBAL/K_GLOBAL 宏；在 main 中解析命令行得到 mt/nt/kt，
//      用运行时变量参与内存分配、初始化、网格尺寸与 kernel 调用。
//      子正规数生成逻辑保持不变，只是改为接收运行时维度。

#include <assert.h>
#include <cstdio>
#include <cstdlib>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <cfloat>    // for FLT_MIN
#include <math.h>    // for ldexpf

// ==== 辅助宏（保留）====
#ifndef MAX
#define MAX(a,b) (( (a) > (b) ) ? (a) : (b))
#endif

#ifndef SHARED_MEMORY_LIMIT_64K
#define SHARED_MEMORY_LIMIT_64K 1
#endif

// GPU 配置
#define WARP_SIZE 32

// WMMA tile 尺寸（固定 16x16x16）
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// 基础 tile 尺寸（与 WMMA 一致）
#define M 16
#define N 16
#define K 16

#define M_SEED 6432

// 实现相关常量（原样保留）
#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#if SHARED_MEMORY_LIMIT_64K
#define CHUNK_K 4
#else
#define CHUNK_K 8
#endif

#define CHUNK_LINE_BYTES (CHUNK_K * K * sizeof(half))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

// 运行时版本不再需要 GLOBAL_MEM_STRIDE 宏
// #define GLOBAL_MEM_STRIDE N_GLOBAL

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)

#define SKEW_HALF 16

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)

using namespace nvcuda;

// ================= 辅助函数 ==================
#define checkCudaErrors(val)  check( (val), #val, __FILE__, __LINE__ )
void check(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\"\n",
                file, line, (int)result, hipGetErrorString(result), func);
        exit(EXIT_FAILURE);
    }
}

// ---- 生成子正规数：half 与 float ----
// half 子正规范围：(2^-24, 2^-14) ≈ (5.96e-8, 6.10e-5)
// 生成一个严格小于 2^-14 的正数，避免转换为 half 时变为 0 或变为 normal
static inline float gen_half_subnormal_pos() {
  const float min_sub = ldexpf(1.0f, -24);            // 2^-24
  const float max_sub = ldexpf(1.0f, -14) * 0.999f;   // < 2^-14
  float u = rand() / (float)RAND_MAX;                 // [0,1]
  return min_sub + u * (max_sub - min_sub);           // (2^-24, 2^-14)
}

// float 子正规：小于 FLT_MIN 的正数
static inline float gen_float_subnormal_pos() {
  // 缩放 FLT_MIN 以确保进入 subnormal 区间
  return (float)(rand() % 100 + 1) * FLT_MIN / 1e5f;  // 典型 ~1e-43
}

// 改：带维度参数（运行时）
__host__ void init_host_matrices(half *a, half *b, float *c,
                                 int M_GLOBAL, int N_GLOBAL, int K_GLOBAL) {
  srand(M_SEED);

  // A: half 子正规
  for (int i = 0; i < M_GLOBAL; i++) {
    for (int j = 0; j < K_GLOBAL; j++) {
      a[i * K_GLOBAL + j] = __float2half(gen_half_subnormal_pos());
    }
  }

  // B: half 子正规
  for (int i = 0; i < N_GLOBAL; i++) {
    for (int j = 0; j < K_GLOBAL; j++) {
      b[i * K_GLOBAL + j] = __float2half(gen_half_subnormal_pos());
    }
  }

  // C: float 子正规
  for (int t = 0; t < M_GLOBAL * N_GLOBAL; t++) {
    c[t] = gen_float_subnormal_pos();
  }
}

// 简单 WMMA GEMM（按运行时维度工作）
__global__ void simple_wmma_gemm(half *a, half *b, float *c, float *d, int m_ld,
                                 int n_ld, int k_ld, float alpha, float beta) {
  int lda = k_ld;
  int ldb = k_ld;
  int ldc = n_ld;

  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  wmma::fill_fragment(acc_frag, 0.0f);

  for (int i = 0; i < k_ld; i += WMMA_K) {
    int aCol = i;
    int aRow = warpM * WMMA_M;
    int bCol = warpN * WMMA_N;  // = 16
    int bRow = i;

    if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
      wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
      wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  int cCol = warpN * WMMA_N;
  int cRow = warpM * WMMA_M;

  if (cRow < m_ld && cCol < n_ld) {
    wmma::load_matrix_sync(c_frag, c + cCol + cRow * ldc, ldc, wmma::mem_row_major);
    for (int i = 0; i < c_frag.num_elements; i++) {
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
    }
    wmma::store_matrix_sync(d + cCol + cRow * ldc, c_frag, ldc, wmma::mem_row_major);
  }
}

int main(int argc, char **argv) {
  // ---- 解析命令行参数：支持 ./gemm s 或 ./gemm mt nt kt ----
  int mt = 2, nt = 2, kt = 2;  // 默认各维 2 个 16x16 tile
  if (argc == 2) {
    int s = atoi(argv[1]);
    if (s > 0) mt = nt = kt = s;
  } else if (argc >= 4) {
    int t1 = atoi(argv[1]);
    int t2 = atoi(argv[2]);
    int t3 = atoi(argv[3]);
    if (t1 > 0) mt = t1;
    if (t2 > 0) nt = t2;
    if (t3 > 0) kt = t3;
  }

  // 运行时全局尺寸（16 的倍数，满足 WMMA 要求）
  const int M_GLOBAL = M * mt;
  const int N_GLOBAL = N * nt;
  const int K_GLOBAL = K * kt;

  // ---- 主机内存 ----
  half  *A_h = (half *)malloc(sizeof(half)  * M_GLOBAL * K_GLOBAL);
  half  *B_h = (half *)malloc(sizeof(half)  * N_GLOBAL * K_GLOBAL);
  float *C_h = (float*)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);
  float *result_hD = (float*)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);

  // ---- 设备内存 ----
  half  *A = NULL;
  half  *B = NULL;
  float *C = NULL;
  float *D = NULL;

  checkCudaErrors(hipMalloc((void**)&A, sizeof(half)  * M_GLOBAL * K_GLOBAL));
  checkCudaErrors(hipMalloc((void**)&B, sizeof(half)  * N_GLOBAL * K_GLOBAL));
  checkCudaErrors(hipMalloc((void**)&C, sizeof(float) * M_GLOBAL * N_GLOBAL));
  checkCudaErrors(hipMalloc((void**)&D, sizeof(float) * M_GLOBAL * N_GLOBAL));

  // ---- 初始化并拷贝 ----
  init_host_matrices(A_h, B_h, C_h, M_GLOBAL, N_GLOBAL, K_GLOBAL);

  checkCudaErrors(hipMemcpy(A, A_h, sizeof(half)  * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(B, B_h, sizeof(half)  * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(C, C_h, sizeof(float) * M_GLOBAL * N_GLOBAL, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(D, 0, sizeof(float) * M_GLOBAL * N_GLOBAL));

  // 共享内存需求（demo 内核未直接使用；保留计算以兼容）
  enum {
    SHMEM_SZ = MAX(
        sizeof(half) * (BLOCK_COL_TILES * M) * (CHUNK_K * K + SKEW_HALF) * 2,
        M * (BLOCK_ROW_WARPS * WARP_ROW_TILES) * N *
            (BLOCK_COL_WARPS * WARP_COL_TILES) * sizeof(float))
  };
  (void)SHMEM_SZ;

  const float alpha = 1.1f;
  const float beta  = 1.2f;

  // ---- 计时 ----
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  checkCudaErrors(hipEventRecord(start));

  // ---- 网格/线程块 ----
  dim3 gridDim, blockDim;
  blockDim.x = 128;  // 必须是 warpSize 的倍数
  blockDim.y = 4;

  gridDim.x = (M_GLOBAL + (WMMA_M * (blockDim.x / 32) - 1)) / (WMMA_M * (blockDim.x / 32));
  gridDim.y = (N_GLOBAL + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

  // ---- kernel ----
  simple_wmma_gemm<<<gridDim, blockDim>>>(A, B, C, D, M_GLOBAL, N_GLOBAL, K_GLOBAL, alpha, beta);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  // ---- D2H ----
  checkCudaErrors(hipMemcpy(result_hD, D, sizeof(float) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost));

  FILE *file = fopen("result.txt", "r");
  if (file == NULL) {
    printf("Failed\n");
    free(A_h); free(B_h); free(C_h); free(result_hD);
    hipFree(A); hipFree(B); hipFree(C); hipFree(D);
    return 0;
  }

  float *expected = (float*) malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);
  int count = 0;
  while (fscanf(file, "%f", &expected[count]) == 1 && count < M_GLOBAL * N_GLOBAL) {
    count++;
  }
  fclose(file);

  if (count != M_GLOBAL * N_GLOBAL) {
    printf("Failed\n");
    free(expected);
    free(A_h); free(B_h); free(C_h); free(result_hD);
    hipFree(A); hipFree(B); hipFree(C); hipFree(D);
    return 0;
  }

  // ===== 显式比较 NaN 和 Inf =====
  bool match = true;
  const float eps = 1e-5f;
  for (int i = 0; i < M_GLOBAL * N_GLOBAL; i++) {
    float actual = result_hD[i];
    float expected_val = expected[i];

    if (isnan(actual) && isnan(expected_val)) continue;
    if (isnan(actual) || isnan(expected_val)) { match = false; break; }

    if (isinf(actual) && isinf(expected_val)) {
      if (signbit(actual) != signbit(expected_val)) { match = false; break; }
      else continue;
    }
    if (isinf(actual) || isinf(expected_val)) { match = false; break; }

    if (fabs(actual - expected_val) > eps) { match = false; break; }
  }

  if (match) {
    printf("Success\n");
  } else {
    printf("Failed\n");
  }

  free(expected);
  free(A_h);
  free(B_h);
  free(C_h);
  free(result_hD);

  checkCudaErrors(hipFree(A));
  checkCudaErrors(hipFree(B));
  checkCudaErrors(hipFree(C));
  checkCudaErrors(hipFree(D));

  return 0;
}
