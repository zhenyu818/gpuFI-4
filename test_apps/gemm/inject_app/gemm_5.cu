#include "hip/hip_runtime.h"
// 用法：
//   nvcc -arch=sm_70 -O3 -o gemm gemm.cu
//   ./gemm s         # M=N=K=16*s
//   ./gemm mt nt kt  # M=16*mt, N=16*nt, K=16*kt

#include <assert.h>
#include <cstdio>
#include <cstdlib>
#include <algorithm>
#include <climits>
#include <cmath>      // for NAN
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

// ==== 辅助宏 ====
#ifndef MAX
#define MAX(a,b) (( (a) > (b) ) ? (a) : (b))
#endif

#ifndef SHARED_MEMORY_LIMIT_64K
#define SHARED_MEMORY_LIMIT_64K 1
#endif

// GPU 配置
#define WARP_SIZE 32

// WMMA tile 尺寸（固定 16x16x16）
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// 基础 tile 尺寸（与 WMMA 一致）
#define M 16
#define N 16
#define K 16

#define M_SEED 6432

// Implementation constants（原样保留）
#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#if SHARED_MEMORY_LIMIT_64K
#define CHUNK_K 4
#else
#define CHUNK_K 8
#endif

#define CHUNK_LINE_BYTES (CHUNK_K * K * sizeof(half))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)

// shared memory bank 冲突移位
#define SKEW_HALF 16

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)

using namespace nvcuda;

// ================= 辅助函数 ==================
#define checkCudaErrors(val)  check( (val), #val, __FILE__, __LINE__ )

void check(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\"\n",
                file, line, (int)result, hipGetErrorString(result), func);
        exit(EXIT_FAILURE);
    }
}

// === 输入生成函数（half/float，包含 NaN） ===
static inline void generate_input_with_special_half(half* buf, int total_len) {
  for (int i = 0; i < total_len; i++) {
    if (rand() % 2 == 0) {
      buf[i] = __float2half(NAN);   // 50% NaN
    } else {
      buf[i] = __float2half((float)(rand() % 10)); // 50% 随机 0–9
    }
  }
}

static inline void generate_input_with_special_float(float* buf, int total_len) {
  for (int i = 0; i < total_len; i++) {
    if (rand() % 2 == 0) {
      buf[i] = NAN;   // 50% NaN
    } else {
      buf[i] = (float)(rand() % 10); // 50% 随机 0–9
    }
  }
}

// 初始化主机端矩阵
__host__ void init_host_matrices(half *a, half *b, float *c,
                                 int M_GLOBAL, int N_GLOBAL, int K_GLOBAL) {
  srand(M_SEED);
  // A: M_GLOBAL x K_GLOBAL
  generate_input_with_special_half(a, M_GLOBAL * K_GLOBAL);
  // B: N_GLOBAL x K_GLOBAL
  generate_input_with_special_half(b, N_GLOBAL * K_GLOBAL);
  // C: M_GLOBAL x N_GLOBAL
  generate_input_with_special_float(c, M_GLOBAL * N_GLOBAL);
}

// 简易 WMMA kernel
__global__ void simple_wmma_gemm(half *a, half *b, float *c, float *d, int m_ld,
                                 int n_ld, int k_ld, float alpha, float beta) {
  int lda = k_ld;
  int ldb = k_ld;
  int ldc = n_ld;

  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  wmma::fill_fragment(acc_frag, 0.0f);

  for (int i = 0; i < k_ld; i += WMMA_K) {
    int aCol = i;
    int aRow = warpM * WMMA_M;
    int bCol = warpN * WMMA_N;
    int bRow = i;

    if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
      wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
      wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  int cCol = warpN * WMMA_N;
  int cRow = warpM * WMMA_M;

  if (cRow < m_ld && cCol < n_ld) {
    wmma::load_matrix_sync(c_frag, c + cCol + cRow * ldc, ldc, wmma::mem_row_major);
    for (int i = 0; i < c_frag.num_elements; i++) {
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
    }
    wmma::store_matrix_sync(d + cCol + cRow * ldc, c_frag, ldc, wmma::mem_row_major);
  }
}

int main(int argc, char **argv) {
  int mt = 2, nt = 2, kt = 2;
  if (argc == 2) {
    int s = atoi(argv[1]);
    if (s > 0) mt = nt = kt = s;
  } else if (argc >= 4) {
    int t1 = atoi(argv[1]);
    int t2 = atoi(argv[2]);
    int t3 = atoi(argv[3]);
    if (t1 > 0) mt = t1;
    if (t2 > 0) nt = t2;
    if (t3 > 0) kt = t3;
  }

  const int M_GLOBAL = M * mt;
  const int N_GLOBAL = N * nt;
  const int K_GLOBAL = K * kt;

  half  *A_h = (half *)malloc(sizeof(half)  * M_GLOBAL * K_GLOBAL);
  half  *B_h = (half *)malloc(sizeof(half)  * N_GLOBAL * K_GLOBAL);
  float *C_h = (float*)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);
  float *result_hD = (float*)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);

  half  *A = NULL;
  half  *B = NULL;
  float *C = NULL;
  float *D = NULL;

  checkCudaErrors(hipMalloc((void**)&A, sizeof(half)  * M_GLOBAL * K_GLOBAL));
  checkCudaErrors(hipMalloc((void**)&B, sizeof(half)  * N_GLOBAL * K_GLOBAL));
  checkCudaErrors(hipMalloc((void**)&C, sizeof(float) * M_GLOBAL * N_GLOBAL));
  checkCudaErrors(hipMalloc((void**)&D, sizeof(float) * M_GLOBAL * N_GLOBAL));

  init_host_matrices(A_h, B_h, C_h, M_GLOBAL, N_GLOBAL, K_GLOBAL);

  checkCudaErrors(hipMemcpy(A, A_h, sizeof(half)  * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(B, B_h, sizeof(half)  * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(C, C_h, sizeof(float) * M_GLOBAL * N_GLOBAL, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(D, 0, sizeof(float) * M_GLOBAL * N_GLOBAL));

  const float alpha = 1.1f;
  const float beta  = 1.2f;

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  checkCudaErrors(hipEventRecord(start));

  dim3 gridDim, blockDim;
  blockDim.x = 128;
  blockDim.y = 4;

  gridDim.x = (M_GLOBAL + (WMMA_M * (blockDim.x / 32) - 1)) / (WMMA_M * (blockDim.x / 32));
  gridDim.y = (N_GLOBAL + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

  simple_wmma_gemm<<<gridDim, blockDim>>>(A, B, C, D, M_GLOBAL, N_GLOBAL, K_GLOBAL, alpha, beta);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMemcpy(result_hD, D, sizeof(float) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost));

  FILE *file = fopen("result.txt", "r");
  if (file == NULL) {
    printf("Failed\n");
    free(A_h); free(B_h); free(C_h); free(result_hD);
    hipFree(A); hipFree(B); hipFree(C); hipFree(D);
    return 0;
  }

  float *expected = (float*) malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);
  int count = 0;
  while (fscanf(file, "%f", &expected[count]) == 1 && count < M_GLOBAL * N_GLOBAL) {
    count++;
  }
  fclose(file);

  if (count != M_GLOBAL * N_GLOBAL) {
    printf("Failed\n");
    free(expected);
    free(A_h); free(B_h); free(C_h); free(result_hD);
    hipFree(A); hipFree(B); hipFree(C); hipFree(D);
    return 0;
  }

  // ===== 显式比较 NaN 和 Inf =====
  bool match = true;
  const float eps = 1e-5f;
  for (int i = 0; i < M_GLOBAL * N_GLOBAL; i++) {
    float actual = result_hD[i];
    float expected_val = expected[i];

    if (isnan(actual) && isnan(expected_val)) continue;
    if (isnan(actual) || isnan(expected_val)) { match = false; break; }

    if (isinf(actual) && isinf(expected_val)) {
      if (signbit(actual) != signbit(expected_val)) { match = false; break; }
      else continue;
    }
    if (isinf(actual) || isinf(expected_val)) { match = false; break; }

    if (fabs(actual - expected_val) > eps) { match = false; break; }
  }

  if (match) {
    printf("Success\n");
  } else {
    printf("Failed\n");
  }

  free(expected);
  free(A_h);
  free(B_h);
  free(C_h);
  free(result_hD);

  checkCudaErrors(hipFree(A));
  checkCudaErrors(hipFree(B));
  checkCudaErrors(hipFree(C));
  checkCudaErrors(hipFree(D));

  return 0;
}
