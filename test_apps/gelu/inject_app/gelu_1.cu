#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>
#include <hip/hip_runtime.h>

#define M_SEED 9182
#define M_BLOCK_SIZE 1024

/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// width is hidden_dim and height is seq_len
__global__ void gelu_bias_loop(float* src, const float* bias, int width, int height)
{
    int batch = blockIdx.x;
    int x     = blockIdx.y;  // seq length
    int y     = threadIdx.x;

    if (x < height) {
        int index = batch * width * height + x * width;

        for (; y < width; y += blockDim.x) {
            float v_src  = src[index + y];
            float v_bias = bias[y];
            float v      = v_src + v_bias;

            // GELU近似公式
            float t = 0.5f * v * (1.0f + tanhf(0.79788456f * (v + 0.044715f * v * v * v)));

            src[index + y] = t;
        }
    }
}

int main(int argc, char* argv[])
{
    if (argc != 4) {
        printf("Usage: %s <batch> <sequence length> <hidden dimension> <repeat>\n", argv[0]);
        return 1;
    }

    const int batch_size = atoi(argv[1]);
    const int seq_len = atoi(argv[2]);
    const int hidden_dim = atoi(argv[3]);
    const int repeat = 1;

    const size_t src_size = (size_t)batch_size * seq_len * hidden_dim;

    const size_t src_size_bytes =  src_size * sizeof(float);
    const int bias_size_bytes = hidden_dim * sizeof(float);

    srand(M_SEED);
    float* output = (float*) malloc (src_size_bytes);
    for (size_t i = 0; i < src_size; i++) {
        output[i] = rand() / (float)RAND_MAX;
    }

    float* bias = (float*) malloc (bias_size_bytes);
    for (int i = 0; i < hidden_dim; i++) {
        bias[i] = -6.0f + (rand() % 12);
    }

    float* d_output;
    hipMalloc((void**)&d_output, src_size_bytes);
    hipMemcpy(d_output, output, src_size_bytes, hipMemcpyHostToDevice);

    float* d_bias;
    hipMalloc((void**)&d_bias, bias_size_bytes);
    hipMemcpy(d_bias, bias, bias_size_bytes, hipMemcpyHostToDevice);
  
    dim3 block(1024, 1);
    dim3 grid(batch_size, seq_len);

    hipDeviceSynchronize();
    for (int i = 0; i < repeat; i++) {
        gelu_bias_loop <<<grid, block>>> (d_output, d_bias, hidden_dim, seq_len);
    }

    hipDeviceSynchronize();

    hipMemcpy(output, d_output, src_size_bytes, hipMemcpyDeviceToHost);

    // ==== 从 result.txt 读取参考值 ====
    FILE* file = fopen("result.txt", "r");
    if (file == NULL) {
        printf("Failed\n");
        hipFree(d_output);
        hipFree(d_bias);
        free(output);
        free(bias);
        return 1;
    }

    float* expected = (float*)malloc(sizeof(float) * src_size);
    int count = 0;
    while (fscanf(file, "%f", &expected[count]) == 1 && count < (int)src_size) {
        count++;
    }
    fclose(file);

    if (count != (int)src_size) {
        printf("Failed\n");
        free(expected);
        hipFree(d_output);
        hipFree(d_bias);
        free(output);
        free(bias);
        return 1;
    }

    // ==== 逐项比较 ====
    bool match = true;
    const float eps = 1e-5f;  // 容许误差
    for (size_t i = 0; i < src_size; i++) {
        if (fabs(output[i] - expected[i]) > eps) {
            match = false;
            break;
        }
    }

    if (match) {
        printf("Success\n");
    } else {
        printf("Failed\n");
    }

    free(expected);
  
    hipFree(d_output);
    hipFree(d_bias);
    free(output);
    free(bias);

    return 0;
}
