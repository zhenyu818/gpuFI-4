#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>
#include <hip/hip_runtime.h>

#define M_SEED 9182
#define M_BLOCK_SIZE 1024

// width is hidden_dim and height is seq_len
__global__ void gelu_bias_loop(float* src, const float* bias, int width, int height)
{
    int batch = blockIdx.x;
    int x     = blockIdx.y;  // seq length
    int y     = threadIdx.x;

    if (x < height) {
        int index = batch * width * height + x * width;

        for (; y < width; y += blockDim.x) {
            float v_src  = src[index + y];
            float v_bias = bias[y];
            float v      = v_src + v_bias;

            // GELU近似公式
            float t = 0.5f * v * (1.0f + tanhf(0.79788456f * (v + 0.044715f * v * v * v)));

            src[index + y] = t;
        }
    }
}

int main(int argc, char* argv[])
{
    if (argc != 4) {
        printf("Usage: %s <batch> <sequence length> <hidden dimension>\n", argv[0]);
        return 1;
    }

    const int batch_size = atoi(argv[1]);
    const int seq_len = atoi(argv[2]);
    const int hidden_dim = atoi(argv[3]);
    const int repeat = 1;

    const size_t src_size = (size_t)batch_size * seq_len * hidden_dim;

    const size_t src_size_bytes =  src_size * sizeof(float);
    const int bias_size_bytes = hidden_dim * sizeof(float);

    srand(M_SEED);
    float* output = (float*) malloc(src_size_bytes);

    // ====== 对抗性输入模式 ======
    for (size_t i = 0; i < src_size; i++) {
        // 模式1：交替极值（正负交替）
        if (i % 2 == 0) {
            output[i] = 10.0f;   // 大正值
        } else {
            output[i] = -10.0f;  // 大负值
        }

        // 模式2（可选）：周期性波动，可替换上面逻辑
        // float val = (i % 100 < 50) ? 5.0f : -5.0f;
        // output[i] = val;
    }

    float* bias = (float*) malloc(bias_size_bytes);
    for (int i = 0; i < hidden_dim; i++) {
        // 让 bias 也有对抗性：大幅度波动
        bias[i] = (i % 2 == 0) ? 6.0f : -6.0f;
    }

    float* d_output;
    hipMalloc((void**)&d_output, src_size_bytes);
    hipMemcpy(d_output, output, src_size_bytes, hipMemcpyHostToDevice);

    float* d_bias;
    hipMalloc((void**)&d_bias, bias_size_bytes);
    hipMemcpy(d_bias, bias, bias_size_bytes, hipMemcpyHostToDevice);
  
    dim3 block(1024, 1);
    dim3 grid(batch_size, seq_len);

    hipDeviceSynchronize();
    for (int i = 0; i < repeat; i++) {
        gelu_bias_loop <<<grid, block>>> (d_output, d_bias, hidden_dim, seq_len);
    }

    hipDeviceSynchronize();

    hipMemcpy(output, d_output, src_size_bytes, hipMemcpyDeviceToHost);

    // ==== 从 result.txt 读取参考值 ====
    FILE* file = fopen("result.txt", "r");
    if (file == NULL) {
        printf("Fault Injection Test Failed!\n");
        hipFree(d_output);
        hipFree(d_bias);
        free(output);
        free(bias);
        return 1;
    }

    float* expected = (float*)malloc(sizeof(float) * src_size);
    int count = 0;
    while (fscanf(file, "%f", &expected[count]) == 1 && count < (int)src_size) {
        count++;
    }
    fclose(file);

    if (count != (int)src_size) {
        printf("Fault Injection Test Failed!\n");
        free(expected);
        hipFree(d_output);
        hipFree(d_bias);
        free(output);
        free(bias);
        return 1;
    }

    // ==== 逐项比较 ====
    bool match = true;
    const float eps = 1e-5f;  // 容许误差
    for (size_t i = 0; i < src_size; i++) {
        if (fabs(output[i] - expected[i]) > eps) {
            match = false;
            break;
        }
    }

    if (match) {
        printf("Fault Injection Test Success!\n");
    } else {
        printf("Fault Injection Test Failed!\n");
    }

    free(expected);
  
    hipFree(d_output);
    hipFree(d_bias);
    free(output);
    free(bias);

    return 0;
}
