#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define RADIUS 7
#define BLOCK_SIZE 256
#define M_SEED 3031

#define SPARSE_N 2 // 每组非零个数
#define SPARSE_M 4 // 每组大小

__global__ void stencil_1d(const int *__restrict__ in, int *__restrict__ out, int length) {
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    if (gindex < length) {
        temp[lindex] = in[gindex];

        if (threadIdx.x < RADIUS) {
            temp[lindex - RADIUS] = (gindex < RADIUS) ? 0 : in[gindex - RADIUS];
            if (gindex + BLOCK_SIZE < length + RADIUS)
                temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
        }

        __syncthreads();

        int result = 0;
        for (int offset = -RADIUS; offset <= RADIUS; offset++)
            result += temp[lindex + offset];

        out[gindex] = result;
    }
}

// ==================== 生成 2:4 稀疏输入 ====================
static void generate_2to4_sparse_group(int *group, int group_len) {
    for (int k = 0; k < group_len; k++)
        group[k] = 0;

    bool selected[SPARSE_M] = {false};
    int selected_count = 0;
    while (selected_count < SPARSE_N) {
        int rand_idx = rand() % SPARSE_M;
        if (!selected[rand_idx]) {
            selected[rand_idx] = true;
            selected_count++;
        }
    }

    for (int k = 0; k < SPARSE_M; k++) {
        if (selected[k]) {
            group[k] = rand() % 10;
            while (group[k] == 0) {
                group[k] = rand() % 10;
            }
        }
    }
}

static void generate_sparse_input(int *a, int length) {
    srand(M_SEED);
    int aligned_len = (length % SPARSE_M == 0) ? length : (length / SPARSE_M + 1) * SPARSE_M;

    for (int i = 0; i < aligned_len; i += SPARSE_M) {
        int group[SPARSE_M];
        generate_2to4_sparse_group(group, SPARSE_M);
        for (int k = 0; k < SPARSE_M; k++) {
            if (i + k < length) {
                a[i + k] = group[k];
            }
        }
    }

    // pad 区域保持随机 (避免访问越界)
    for (int i = length; i < length + RADIUS; i++) {
        a[i] = rand() % 10;
    }
}
// =========================================================

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s <length>\n", argv[0]);
        printf("length is a multiple of %d\n", BLOCK_SIZE);
        return 1;
    }
    const int length = atoi(argv[1]);
    const int repeat = 1;

    int size = length * sizeof(int);
    int pad_size = (length + RADIUS) * sizeof(int);

    int *a, *b;
    a = (int *)malloc(pad_size);
    b = (int *)malloc(size);

    // 使用 2:4 稀疏生成方法
    generate_sparse_input(a, length);

    int *d_a, *d_b;
    hipMalloc((void **)&d_a, pad_size);
    hipMalloc((void **)&d_b, size);

    hipMemcpy(d_a, a, pad_size, hipMemcpyHostToDevice);

    dim3 grids(length / BLOCK_SIZE);
    dim3 blocks(BLOCK_SIZE);

    hipDeviceSynchronize();

    for (int i = 0; i < repeat; i++)
        stencil_1d<<<grids, blocks>>>(d_a, d_b, length);

    hipDeviceSynchronize();

    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

    // ===== 比对结果 =====
    FILE *file = fopen("result.txt", "r");
    if (file == NULL) {
        printf("Fault Injection Test Failed!\n");
        free(a);
        free(b);
        hipFree(d_a);
        hipFree(d_b);
        return 0;
    }

    int *expected = (int *)malloc(size);
    int i = 0;
    while (fscanf(file, "%d", &expected[i]) == 1 && i < length) {
        i++;
    }
    fclose(file);

    if (i != length) {
        printf("Fault Injection Test Failed!\n");
    } else {
        bool match = true;
        for (i = 0; i < length; i++) {
            if (b[i] != expected[i]) {
                match = false;
                break;
            }
        }
        if (match) {
            printf("Fault Injection Test Success!\n");
        } else {
            printf("Fault Injection Test Failed!\n");
        }
    }

    free(expected);
    free(a);
    free(b);
    hipFree(d_a);
    hipFree(d_b);
    return 0;
}
