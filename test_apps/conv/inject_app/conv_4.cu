/*
  Reference
  Chapter 7 in Programming massively parallel processors,
  A hands-on approach (D. Kirk and W. Hwu)
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

#define M_SEED 6498
#define BLOCK_SIZE 256
#define MAX_MASK_WIDTH 10
#define TILE_SIZE BLOCK_SIZE
#define SPARSE_N 2
#define SPARSE_M 4

template<typename T>
__constant__ T mask [MAX_MASK_WIDTH];

template<typename T>
__global__
void conv1d_tiled_caching(const T *__restrict__ in,
                                T *__restrict__ out,
                          const int input_width,
                          const int mask_width)
{
  __shared__ T tile[TILE_SIZE];

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  tile[threadIdx.x] = in[i];
  __syncthreads();

  int this_tile_start = blockIdx.x * blockDim.x;
  int next_tile_start = (blockIdx.x + 1) * blockDim.x;
  int start = i - (mask_width / 2);
  T s = 0;
  for (int j = 0; j < mask_width; j++) {
    int in_index = start + j;
    if (in_index >= 0 && in_index < input_width) {
      if (in_index >= this_tile_start && in_index < next_tile_start) {
        s += tile[threadIdx.x + j - (mask_width / 2)] * mask<T>[j];
      } else {
        s += in[in_index] * mask<T>[j];
      }
    }
  }
  out[i] = s;
}

/**
 * ---- 新增: 2:4 稀疏输入生成 ----
 */
template<typename T>
static void generate_2to4_sparse(T *data, int size) {
  srand(M_SEED);
  for (int i = 0; i < size; i += SPARSE_M) {
    bool selected[SPARSE_M] = {false};
    int selected_count = 0;
    // 随机挑 2 个位置保留非零
    while (selected_count < SPARSE_N) {
      int idx = rand() % SPARSE_M;
      if (!selected[idx]) {
        selected[idx] = true;
        selected_count++;
      }
    }
    for (int k = 0; k < SPARSE_M && i + k < size; k++) {
      if (selected[k]) {
        data[i + k] = (T)((rand() % 10) + 1);  // 非零范围 [1,10]
      } else {
        data[i + k] = (T)0;
      }
    }
  }
}

template <typename T>
void conv1D(const int input_width, const int mask_width, const int repeat)
{
  size_t size_bytes = input_width * sizeof(T);

  T *a, *b;
  a = (T *)malloc(size_bytes); // input
  b = (T *)malloc(size_bytes); // output

  T h_mask[MAX_MASK_WIDTH];
  for (int i = 0; i < MAX_MASK_WIDTH; i++) h_mask[i] = 1; 

  // ---- 修改: 使用 2:4 稀疏初始化 ----
  generate_2to4_sparse<T>(a, input_width);

  T *d_a, *d_b;
  hipMalloc((void **)&d_a, size_bytes);
  hipMalloc((void **)&d_b, size_bytes);

  hipMemcpy(d_a, a, size_bytes, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(mask<T>), h_mask, mask_width * sizeof(T));

  dim3 grids (input_width / BLOCK_SIZE);
  dim3 blocks (BLOCK_SIZE);

  hipDeviceSynchronize();

  // conv1D tiling and caching
  for (int i = 0; i < repeat; i++) {
    conv1d_tiled_caching <<< grids, blocks >>> (d_a, d_b, input_width, mask_width);
  }
  hipDeviceSynchronize();
  hipMemcpy(b, d_b, size_bytes, hipMemcpyDeviceToHost);
  // ===== 从 result.txt 读取参考结果 =====
  FILE *file = fopen("result.txt", "r");
  if (file == NULL) {
    printf("Fault Injection Test Failed!\n");
    free(a); free(b);
    hipFree(d_a); hipFree(d_b);
    return;
  }

  T *expected = (T*) malloc(size_bytes);
  int count = 0;
  while (fscanf(file, "%hd", &expected[count]) == 1 && count < input_width) {
    count++;
  }
  fclose(file);

  if (count != input_width) {
    printf("Fault Injection Test Failed!\n");
    free(expected);
    free(a); free(b);
    hipFree(d_a); hipFree(d_b);
    return;
  }

  // ===== 逐项比较结果 =====
  bool match = true;
  for (int i = 0; i < input_width; i++) {
    if (b[i] != expected[i]) {
      match = false;
      break;
    }
  }

  if (match) {
    printf("Fault Injection Test Success!\n");
  } else {
    printf("Fault Injection Test Failed!\n");
  }

  free(expected);

  free(a);
  free(b);
  hipFree(d_a);
  hipFree(d_b);
}

int main(int argc, char* argv[]) {
  if (argc != 3) {
    printf("Usage: %s <input_width> <mask_width>\n", argv[0]);
    return 1;
  }

  int input_width = atoi(argv[1]);
  // a multiple of BLOCK_SIZE
  input_width = (input_width + BLOCK_SIZE - 1) / BLOCK_SIZE * BLOCK_SIZE;
  
  const int mask_width = atoi(argv[2]);

  const int repeat = 1;

  conv1D<int16_t>(input_width, mask_width, repeat);

  return 0;
}
