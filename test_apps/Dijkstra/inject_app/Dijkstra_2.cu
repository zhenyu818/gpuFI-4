// Inject app for Dijkstra Kernel1 (all-zero weights)
#include <hip/hip_runtime.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define D_SEED 1111
#define BLOCK_SIZE 256

#define CUDA_CHECK(call)                                                                                               \
    do {                                                                                                               \
        hipError_t err__ = (call);                                                                                    \
        if (err__ != hipSuccess) {                                                                                    \
            fprintf(stderr, "CUDA error at %s:%d code=%d(%s)\n", __FILE__, __LINE__, (int)err__,                       \
                    hipGetErrorString(err__));                                                                        \
            exit(EXIT_FAILURE);                                                                                        \
        }                                                                                                              \
    } while (0)

static inline int iDivUp(int a, int b) { return (a + b - 1) / b; }

__device__ inline float atomicMinFloat(float *addr, float value) {
    float old = __int_as_float(atomicCAS((int *)addr, __float_as_int(*addr), __float_as_int(*addr)));
    while (value < old) {
        float assumed = old;
        int old_int = atomicCAS((int *)addr, __float_as_int(assumed), __float_as_int(value));
        old = __int_as_float(old_int);
        if (old == assumed)
            break;
    }
    return old;
}
__device__ inline float atomicMin(float *addr, float value) { return atomicMinFloat(addr, value); }

__global__ void Kernel1(const int *__restrict__ vertexArray, const int *__restrict__ edgeArray,
                        const float *__restrict__ weightArray, bool *__restrict__ finalizedVertices,
                        float *__restrict__ shortestDistances, float *__restrict__ updatingShortestDistances,
                        const int numVertices, const int numEdges) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numVertices) {

        if (finalizedVertices[tid] == true) {

            finalizedVertices[tid] = false;

            int edgeStart = vertexArray[tid], edgeEnd;

            if (tid + 1 < (numVertices))
                edgeEnd = vertexArray[tid + 1];
            else
                edgeEnd = numEdges;

            for (int edge = edgeStart; edge < edgeEnd; edge++) {
                int nid = edgeArray[edge];
                atomicMin(&updatingShortestDistances[nid], shortestDistances[tid] + weightArray[edge]);
            }
        }
    }
}

static void generate_graph(int V, int deg, int *vertexArray, int *edgeArray) {
    for (int i = 0; i < V; ++i) vertexArray[i] = i * deg;
    for (int u = 0; u < V; ++u) {
        for (int j = 0; j < deg; ++j) {
            int v;
            int tries = 0;
            for (;;) {
                v = rand() % V;
                if (v == u) {
                    if (++tries < 1000) continue;
                }
                bool dup = false;
                for (int k = 0; k < j; ++k) {
                    if (edgeArray[u * deg + k] == v) {
                        dup = true;
                        break;
                    }
                }
                if (!dup && v != u) break;
            }
            edgeArray[u * deg + j] = v;
        }
    }
}

int main(int argc, char **argv) {
    int V = 64;
    int DEG = 4;
    int src = 0;

    if (argc >= 3) {
        V = atoi(argv[1]);
        DEG = atoi(argv[2]);
        if (V <= 0 || DEG <= 0) return 0;
        if (DEG > V - 1) DEG = V - 1;
    }

    srand(D_SEED);

    const int E = V * DEG;

    int *h_vertexArray = (int *)malloc(sizeof(int) * V);
    int *h_edgeArray = (int *)malloc(sizeof(int) * E);
    float *h_weightArray = (float *)malloc(sizeof(float) * E);
    bool *h_finalizedVertices = (bool *)malloc(sizeof(bool) * V);
    float *h_shortestDistances = (float *)malloc(sizeof(float) * V);
    float *h_updatingShortestDistances = (float *)malloc(sizeof(float) * V);

    generate_graph(V, DEG, h_vertexArray, h_edgeArray);
    for (int i = 0; i < E; ++i) {
        h_weightArray[i] = 0.0f;
    }

    for (int i = 0; i < V; ++i) {
        h_finalizedVertices[i] = (i == src);
        h_shortestDistances[i] = (i == src) ? 0.0f : FLT_MAX;
        h_updatingShortestDistances[i] = h_shortestDistances[i];
    }

    int *d_vertexArray = nullptr, *d_edgeArray = nullptr;
    float *d_weightArray = nullptr, *d_shortestDistances = nullptr, *d_updatingShortestDistances = nullptr;
    bool *d_finalizedVertices = nullptr;

    CUDA_CHECK(hipMalloc(&d_vertexArray, sizeof(int) * V));
    CUDA_CHECK(hipMalloc(&d_edgeArray, sizeof(int) * E));
    CUDA_CHECK(hipMalloc(&d_weightArray, sizeof(float) * E));
    CUDA_CHECK(hipMalloc(&d_finalizedVertices, sizeof(bool) * V));
    CUDA_CHECK(hipMalloc(&d_shortestDistances, sizeof(float) * V));
    CUDA_CHECK(hipMalloc(&d_updatingShortestDistances, sizeof(float) * V));

    CUDA_CHECK(hipMemcpy(d_vertexArray, h_vertexArray, sizeof(int) * V, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_edgeArray, h_edgeArray, sizeof(int) * E, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_weightArray, h_weightArray, sizeof(float) * E, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_finalizedVertices, h_finalizedVertices, sizeof(bool) * V, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_shortestDistances, h_shortestDistances, sizeof(float) * V, hipMemcpyHostToDevice));
    CUDA_CHECK(
        hipMemcpy(d_updatingShortestDistances, h_updatingShortestDistances, sizeof(float) * V, hipMemcpyHostToDevice));

    dim3 block(BLOCK_SIZE);
    dim3 grid(iDivUp(V, BLOCK_SIZE));
    Kernel1<<<grid, block>>>(d_vertexArray, d_edgeArray, d_weightArray, d_finalizedVertices, d_shortestDistances,
                             d_updatingShortestDistances, V, E);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(
        hipMemcpy(h_updatingShortestDistances, d_updatingShortestDistances, sizeof(float) * V, hipMemcpyDeviceToHost));

    FILE *f = fopen("result.txt", "r");
    if (!f) {
        printf("Fault Injection Test Failed!\n");
        return 0;
    }
    float *expected = (float *)malloc(sizeof(float) * V);
    int count = 0;
    while (count < V && fscanf(f, "%f", &expected[count]) == 1) count++;
    fclose(f);
    if (count != V) {
        printf("Fault Injection Test Failed!\n");
        free(expected);
        return 0;
    }
    const float eps = 1e-5f;
    bool ok = true;
    for (int i = 0; i < V; ++i) {
        float a = h_updatingShortestDistances[i];
        float b = expected[i];
        if (isnan(a) && isnan(b)) continue;
        if (isnan(a) || isnan(b)) { ok = false; break; }
        if (isinf(a) && isinf(b)) continue;
        if (isinf(a) || isinf(b)) { ok = false; break; }
        if (fabsf(a - b) > eps) { ok = false; break; }
    }
    if (ok)
        printf("Fault Injection Test Success!\n");
    else
        printf("Fault Injection Test Failed!\n");
    free(expected);

    free(h_vertexArray);
    free(h_edgeArray);
    free(h_weightArray);
    free(h_finalizedVertices);
    free(h_shortestDistances);
    free(h_updatingShortestDistances);
    hipFree(d_vertexArray);
    hipFree(d_edgeArray);
    hipFree(d_weightArray);
    hipFree(d_finalizedVertices);
    hipFree(d_shortestDistances);
    hipFree(d_updatingShortestDistances);
    return 0;
}

