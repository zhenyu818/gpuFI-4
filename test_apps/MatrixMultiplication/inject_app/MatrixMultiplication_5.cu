#include <assert.h>
#include <climits>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define M_SEED 1745
#define M_BLOCK_SIZE 16

// ================= 辅助函数 ==================
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line, (unsigned int)result,
                hipGetErrorString(result), func);
        exit(EXIT_FAILURE);
    }
}

bool checkCmdLineFlag(int argc, const char **argv, const char *flag) {
    for (int i = 0; i < argc; i++) {
        if (!strcmp(argv[i], flag))
            return true;
    }
    return false;
}

int getCmdLineArgumentInt(int argc, const char **argv, const char *argName) {
    for (int i = 0; i < argc - 1; i++) {
        if (!strcmp(argv[i], argName)) {
            return atoi(argv[i + 1]);
        }
    }
    return 0;
}

// ================= CUDA Kernel ==================
template <int BLOCK_SIZE> __global__ void MatrixMulCUDA(double *C, const double *A, const double *B, int wA, int wB) {
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int aBegin = wA * BLOCK_SIZE * by;
    int aEnd = aBegin + wA - 1;
    int aStep = BLOCK_SIZE;

    int bBegin = BLOCK_SIZE * bx;
    int bStep = BLOCK_SIZE * wB;

    double Csub = 0.0;

    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        __syncthreads();

#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }

        __syncthreads();
    }

    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

// ================= 输入生成函数（包含 NaN） ==================
void RandomInitSpecial(double *data, int size) {
    srand(M_SEED);
    for (int i = 0; i < size; i++) {
        int rand_val = rand() % 100;
        if (rand_val < 50) {
            data[i] = NAN; // 50% 概率 NaN
        } else {
            data[i] = (double)(rand() % 10); // 50% 概率 0–9
        }
    }
}

// ================= 主计算函数 ==================
int MatrixMultiply(const dim3 &dimsA, const dim3 &dimsB) {
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(double) * size_A;
    double *h_A;
    checkCudaErrors(hipHostMalloc(&h_A, mem_size_A, hipHostMallocDefault));

    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(double) * size_B;
    double *h_B;
    checkCudaErrors(hipHostMalloc(&h_B, mem_size_B, hipHostMallocDefault));

    RandomInitSpecial(h_A, size_A);
    RandomInitSpecial(h_B, size_B);

    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = sizeof(double) * dimsC.x * dimsC.y;
    double *h_C;
    checkCudaErrors(hipHostMalloc(&h_C, mem_size_C, hipHostMallocDefault));

    double *d_A, *d_B, *d_C;
    checkCudaErrors(hipMalloc((void **)&d_A, mem_size_A));
    checkCudaErrors(hipMalloc((void **)&d_B, mem_size_B));
    checkCudaErrors(hipMalloc((void **)&d_C, mem_size_C));

    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

    dim3 threads(M_BLOCK_SIZE, M_BLOCK_SIZE);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    MatrixMulCUDA<M_BLOCK_SIZE><<<grid, threads>>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

    // ====== 从 result.txt 读取期望结果 ======
    FILE *file = fopen("result.txt", "r");
    if (file == NULL) {
        printf("Fault Injection Test Failed!\n");
        checkCudaErrors(hipHostFree(h_A));
        checkCudaErrors(hipHostFree(h_B));
        checkCudaErrors(hipHostFree(h_C));
        checkCudaErrors(hipFree(d_A));
        checkCudaErrors(hipFree(d_B));
        checkCudaErrors(hipFree(d_C));
        return 0;
    }

    double *expected = (double *)malloc(mem_size_C);
    int count = 0;
    while (fscanf(file, "%lf", &expected[count]) == 1 && count < (int)(dimsC.x * dimsC.y)) {
        count++;
    }
    fclose(file);

    if (count != (int)(dimsC.x * dimsC.y)) {
        printf("Fault Injection Test Failed!\n");
        free(expected);
        checkCudaErrors(hipHostFree(h_A));
        checkCudaErrors(hipHostFree(h_B));
        checkCudaErrors(hipHostFree(h_C));
        checkCudaErrors(hipFree(d_A));
        checkCudaErrors(hipFree(d_B));
        checkCudaErrors(hipFree(d_C));
        return 0;
    }

    // ====== 逐项比较结果 ======
    bool match = true;
    const double eps = 1e-12;
    for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++) {
        double actual = h_C[i];
        double expected_val = expected[i];

        if (isnan(actual) && isnan(expected_val))
            continue;
        if (isnan(actual) || isnan(expected_val)) {
            match = false;
            break;
        }

        if (isinf(actual) && isinf(expected_val)) {
            if (signbit(actual) != signbit(expected_val)) {
                match = false;
                break;
            } else
                continue;
        }
        if (isinf(actual) || isinf(expected_val)) {
            match = false;
            break;
        }

        if (fabs(actual - expected_val) > eps) {
            match = false;
            break;
        }
    }

    if (match)
        printf("Fault Injection Test Success!\n");
    else
        printf("Fault Injection Test Failed!\n");

    free(expected);
    checkCudaErrors(hipHostFree(h_A));
    checkCudaErrors(hipHostFree(h_B));
    checkCudaErrors(hipHostFree(h_C));
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    return 0;
}

// ================= 主函数 ==================
int main(int argc, char **argv) {
    int block_size = M_BLOCK_SIZE;

    dim3 dimsA(5 * 2 * block_size, 5 * 2 * block_size, 1);
    dim3 dimsB(5 * 4 * block_size, 5 * 2 * block_size, 1);

    if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
        dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
    }
    if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
        dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
    }
    if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
        dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
    }
    if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
        dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
    }

    if (dimsA.x != dimsB.y) {
        fprintf(stderr, "Error: Matrix dimensions do not match for multiplication!\n");
        exit(EXIT_FAILURE);
    }

    MatrixMultiply(dimsA, dimsB);
    return 0;
}