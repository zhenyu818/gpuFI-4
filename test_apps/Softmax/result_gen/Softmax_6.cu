#include <cfloat> // for FLT_MIN
#include <chrono>
#include <climits>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256
#define M_SEED 3608

__global__ void softMax(const int numSlice, const int sliceSize, const float *src, float *dest) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numSlice)
        return;

    float max_ = src[i * sliceSize];
    for (int j = 0; j < sliceSize; j++) {
        max_ = max(max_, src[i * sliceSize + j]);
    }

    float sum = 0;
    for (int j = 0; j < sliceSize; j++) {
        sum += expf(src[i * sliceSize + j] - max_);
    }

    for (int j = 0; j < sliceSize; j++) {
        dest[i * sliceSize + j] = expf(src[i * sliceSize + j] - max_) / sum;
    }
}

// ---- 输入生成函数：只生成非正规数 ----
static void generate_input_subnormal(float *input, int numSlice, int sliceSize) {
    srand(M_SEED);
    for (int i = 0; i < numSlice; i++) {
        for (int j = 0; j < sliceSize; j++) {
            // 生成比 FLT_MIN 更小的正数，属于 subnormal
            float tiny = (float)(rand() % 100 + 1) * FLT_MIN / 1e5f;
            input[i * sliceSize + j] = tiny;
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s <number of slices> <slice size>\n", argv[0]);
        return 1;
    }

    int numSlice = atoi(argv[1]);
    int sliceSize = atoi(argv[2]);
    int repeat = 1;
    int numElem = numSlice * sliceSize;

    float *input = (float *)aligned_alloc(1024, sizeof(float) * numElem);
    float *output_gpu = (float *)aligned_alloc(1024, sizeof(float) * numElem);

    // ---- 使用非正规数输入生成 ----
    generate_input_subnormal(input, numSlice, sliceSize);

    float *d_input, *d_output;
    hipMalloc((void **)&d_input, sizeof(float) * numElem);
    hipMalloc((void **)&d_output, sizeof(float) * numElem);
    hipMemcpy(d_input, input, sizeof(float) * numElem, hipMemcpyHostToDevice);

    dim3 global_work_size((numSlice + BLOCK_SIZE - 1) / BLOCK_SIZE * BLOCK_SIZE);
    dim3 local_work_size(BLOCK_SIZE);

    hipDeviceSynchronize();

    for (int n = 0; n < repeat; n++) {
        softMax<<<global_work_size, local_work_size>>>(numSlice, sliceSize, d_input, d_output);
    }

    hipDeviceSynchronize();

    hipMemcpy(output_gpu, d_output, sizeof(float) * numElem, hipMemcpyDeviceToHost);

    // ---------------- 直接输出结果 ----------------
    for (int i = 0; i < numElem; i++) {
        printf("%.6f%c", output_gpu[i], (i == numElem - 1) ? '\n' : ' ');
    }

    free(input);
    free(output_gpu);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
