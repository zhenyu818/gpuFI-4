#include <chrono>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>

#define BLOCK_SIZE 256
#define M_SEED 3608

__global__
void softMax(const int numSlice, const int sliceSize,
             const float* src, float* dest)
{
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= numSlice) return;

  float max_ = src[i * sliceSize];
  for (int j = 0; j < sliceSize; j++) {
    max_ = max(max_, src[i * sliceSize + j]);
  }

  float sum = 0;
  for (int j = 0; j < sliceSize; j++) {
    sum += expf(src[i * sliceSize + j] - max_);
  }

  for (int j = 0; j < sliceSize; j++) {
    dest[i * sliceSize + j] = expf(src[i * sliceSize + j] - max_) / sum;
  }
}

int main(int argc, char* argv[]) {
  if (argc != 3) {
    printf("Usage: %s <number of slices> <slice size>\n", argv[0]);
    return 1;
  }
   
  int numSlice = atoi(argv[1]);
  int sliceSize = atoi(argv[2]);
  int repeat = 1;
  int numElem = numSlice * sliceSize;

  float* input = (float*) aligned_alloc(1024, sizeof(float) * numElem);
  float* output_gpu = (float*) aligned_alloc(1024, sizeof(float) * numElem);

  srand(M_SEED);
  for (int i = 0; i < numSlice; i++)
    for (int j = 0; j < sliceSize; j++)
      input[i*sliceSize+j] = 1; 

  float *d_input, *d_output;
  hipMalloc((void**)&d_input, sizeof(float) * numElem);
  hipMalloc((void**)&d_output, sizeof(float) * numElem);
  hipMemcpy(d_input, input, sizeof(float) * numElem, hipMemcpyHostToDevice);

  dim3 global_work_size ((numSlice+BLOCK_SIZE-1)/BLOCK_SIZE*BLOCK_SIZE);
  dim3 local_work_size (BLOCK_SIZE);

  hipDeviceSynchronize();

  for (int n = 0; n < repeat; n++) {
    softMax<<<global_work_size, local_work_size>>>(numSlice, sliceSize, d_input, d_output);
  }

  hipDeviceSynchronize();

  hipMemcpy(output_gpu, d_output, sizeof(float) * numElem, hipMemcpyDeviceToHost);

  // ---------------- 直接输出结果 ----------------
  for (int i = 0; i < numElem; i++) {
      printf("%.6f%c", output_gpu[i], (i == numElem - 1) ? '\n' : ' ');
  }

  free(input);
  free(output_gpu);
  hipFree(d_input);
  hipFree(d_output);

  return 0;
}
