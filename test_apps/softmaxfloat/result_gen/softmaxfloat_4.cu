#include <chrono>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdbool.h>

#define BLOCK_SIZE 256
#define M_SEED 3608
#define SPARSE_N 2    // 每组保留的非零数
#define SPARSE_M 4    // 每组的元素个数（2:4 稀疏）

__global__
void softMax(const int numSlice, const int sliceSize,
             const float* src, float* dest)
{
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= numSlice) return;

  float max_ = src[i * sliceSize];
  for (int j = 0; j < sliceSize; j++) {
    max_ = max(max_, src[i * sliceSize + j]);
  }

  float sum = 0;
  for (int j = 0; j < sliceSize; j++) {
    sum += expf(src[i * sliceSize + j] - max_);
  }

  for (int j = 0; j < sliceSize; j++) {
    dest[i * sliceSize + j] = expf(src[i * sliceSize + j] - max_) / sum;
  }
}

// ---- 生成 2:4 稀疏分组 ----
static void generate_2to4_sparse_group(float *group, int group_len) {
  for (int k = 0; k < group_len; k++) group[k] = 0.0f;

  bool selected[SPARSE_M] = {false};
  int selected_count = 0;
  while (selected_count < SPARSE_N) {
    int idx = rand() % SPARSE_M;
    if (!selected[idx]) {
      selected[idx] = true;
      selected_count++;
    }
  }

  for (int k = 0; k < group_len; k++) {
    if (selected[k]) {
      group[k] = static_cast<float>(rand() % 10);
      while (group[k] == 0.0f) {
        group[k] = static_cast<float>(rand() % 10);
      }
    }
  }
}

int main(int argc, char* argv[]) {
  if (argc != 3) {
    printf("Usage: %s <number of slices> <slice size>\n", argv[0]);
    return 1;
  }
   
  int numSlice = atoi(argv[1]);
  int sliceSize = atoi(argv[2]);
  int repeat = 1;
  int numElem = numSlice * sliceSize;

  // ---- 确保 sliceSize 为 4 的倍数 ----
  if (sliceSize % SPARSE_M != 0) {
    int new_size = (sliceSize / SPARSE_M + 1) * SPARSE_M;
    printf("Warning: sliceSize (%d) not multiple of %d, auto-adjust to %d\n",
           sliceSize, SPARSE_M, new_size);
    sliceSize = new_size;
    numElem = numSlice * sliceSize;
  }

  float* input = (float*) aligned_alloc(1024, sizeof(float) * numElem);
  float* output_gpu = (float*) aligned_alloc(1024, sizeof(float) * numElem);

  srand(M_SEED);
  for (int i = 0; i < numSlice; i++) {
    for (int j = 0; j < sliceSize; j += SPARSE_M) {
      float group[SPARSE_M];
      generate_2to4_sparse_group(group, SPARSE_M);
      for (int k = 0; k < SPARSE_M; k++) {
        input[i * sliceSize + j + k] = group[k];
      }
    }
  }

  float *d_input, *d_output;
  hipMalloc((void**)&d_input, sizeof(float) * numElem);
  hipMalloc((void**)&d_output, sizeof(float) * numElem);
  hipMemcpy(d_input, input, sizeof(float) * numElem, hipMemcpyHostToDevice);

  dim3 global_work_size ((numSlice+BLOCK_SIZE-1)/BLOCK_SIZE*BLOCK_SIZE);
  dim3 local_work_size (BLOCK_SIZE);

  hipDeviceSynchronize();

  for (int n = 0; n < repeat; n++) {
    softMax<<<global_work_size, local_work_size>>>(numSlice, sliceSize, d_input, d_output);
  }

  hipDeviceSynchronize();

  hipMemcpy(output_gpu, d_output, sizeof(float) * numElem, hipMemcpyDeviceToHost);

  // ---------------- 直接输出结果 ----------------
  for (int i = 0; i < numElem; i++) {
      printf("%.6f%c", output_gpu[i], (i == numElem - 1) ? '\n' : ' ');
  }

  free(input);
  free(output_gpu);
  hipFree(d_input);
  hipFree(d_output);

  return 0;
}
