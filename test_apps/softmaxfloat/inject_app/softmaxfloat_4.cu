#include <chrono>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdbool.h>

#define BLOCK_SIZE 256
#define M_SEED 3608
#define SPARSE_N 2    // 每组保留的非零数
#define SPARSE_M 4    // 每组的元素个数（2:4 稀疏）

__global__
void softMax(const int numSlice, const int sliceSize,
             const float* src, float* dest)
{
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= numSlice) return;

  float max_ = src[i * sliceSize];
  for (int j = 0; j < sliceSize; j++) {
    max_ = max(max_, src[i * sliceSize + j]);
  }

  float sum = 0;
  for (int j = 0; j < sliceSize; j++) {
    sum += expf(src[i * sliceSize + j] - max_);
  }

  for (int j = 0; j < sliceSize; j++) {
    dest[i * sliceSize + j] = expf(src[i * sliceSize + j] - max_) / sum;
  }
}

// ---- 生成 2:4 稀疏分组 ----
static void generate_2to4_sparse_group(float *group, int group_len) {
  for (int k = 0; k < group_len; k++) group[k] = 0.0f;

  bool selected[SPARSE_M] = {false};
  int selected_count = 0;
  while (selected_count < SPARSE_N) {
    int idx = rand() % SPARSE_M;
    if (!selected[idx]) {
      selected[idx] = true;
      selected_count++;
    }
  }

  for (int k = 0; k < group_len; k++) {
    if (selected[k]) {
      group[k] = static_cast<float>(rand() % 10);
      while (group[k] == 0.0f) {
        group[k] = static_cast<float>(rand() % 10);
      }
    }
  }
}

int main(int argc, char* argv[]) {
  if (argc != 3) {
    printf("Usage: %s <number of slices> <slice size>\n", argv[0]);
    return 1;
  }
   
  int numSlice = atoi(argv[1]);
  int sliceSize = atoi(argv[2]);
  int repeat = 1;
  int numElem = numSlice * sliceSize;

  // ---- 确保 sliceSize 为 4 的倍数 ----
  if (sliceSize % SPARSE_M != 0) {
    int new_size = (sliceSize / SPARSE_M + 1) * SPARSE_M;
    printf("Warning: sliceSize (%d) not multiple of %d, auto-adjust to %d\n",
           sliceSize, SPARSE_M, new_size);
    sliceSize = new_size;
    numElem = numSlice * sliceSize;
  }

  float* input = (float*) aligned_alloc(1024, sizeof(float) * numElem);
  float* output_gpu = (float*) aligned_alloc(1024, sizeof(float) * numElem);

  srand(M_SEED);
  for (int i = 0; i < numSlice; i++) {
    for (int j = 0; j < sliceSize; j += SPARSE_M) {
      float group[SPARSE_M];
      generate_2to4_sparse_group(group, SPARSE_M);
      for (int k = 0; k < SPARSE_M; k++) {
        input[i * sliceSize + j + k] = group[k];
      }
    }
  }

  float *d_input, *d_output;
  hipMalloc((void**)&d_input, sizeof(float) * numElem);
  hipMalloc((void**)&d_output, sizeof(float) * numElem);
  hipMemcpy(d_input, input, sizeof(float) * numElem, hipMemcpyHostToDevice);

  dim3 global_work_size ((numSlice+BLOCK_SIZE-1)/BLOCK_SIZE*BLOCK_SIZE);
  dim3 local_work_size (BLOCK_SIZE);

  hipDeviceSynchronize();

  for (int n = 0; n < repeat; n++) {
    softMax<<<global_work_size, local_work_size>>>(numSlice, sliceSize, d_input, d_output);
  }

  hipDeviceSynchronize();

  hipMemcpy(output_gpu, d_output, sizeof(float) * numElem, hipMemcpyDeviceToHost);

  // ===== 从 result.txt 读取期望结果 =====
  FILE *file = fopen("result.txt", "r");
  if (file == NULL) {
    printf("Failed (cannot open result.txt)\n");
    free(input);
    free(output_gpu);
    hipFree(d_input);
    hipFree(d_output);
    return 1;
  }

  float *expected = (float*) malloc(sizeof(float) * numElem);
  int count = 0;
  while (fscanf(file, "%f", &expected[count]) == 1 && count < numElem) {
    count++;
  }
  fclose(file);

  if (count != numElem) {
    printf("Failed (result.txt does not match expected size)\n");
    free(input);
    free(output_gpu);
    free(expected);
    hipFree(d_input);
    hipFree(d_output);
    return 1;
  }

  // ===== 逐项比对结果 =====
  bool match = true;
  const float eps = 1e-4;
  for (int i = 0; i < numElem; i++) {
    if (fabs(output_gpu[i] - expected[i]) > eps) {
      match = false;
      break;
    }
  }

  if (match) {
    printf("Success\n");
  } else {
    printf("Failed\n");
  }

  free(input);
  free(output_gpu);
  free(expected);
  hipFree(d_input);
  hipFree(d_output);

  return 0;
}
