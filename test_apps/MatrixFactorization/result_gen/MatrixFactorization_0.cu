#include <hip/hip_runtime.h>

#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <limits>
#include <random>
#include <vector>

#define HOST_RANDOM_SEED 4124

#define CUDA_CHECK(call)                                                                                               \
    do {                                                                                                               \
        hipError_t err__ = (call);                                                                                    \
        if (err__ != hipSuccess) {                                                                                    \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " code=" << err__ << " ("                  \
                      << hipGetErrorString(err__) << ")\n";                                                           \
            std::exit(EXIT_FAILURE);                                                                                   \
        }                                                                                                              \
    } while (0)

namespace config {
__constant__ int n_factors;
__constant__ float learning_rate;
__constant__ float P_reg;
__constant__ float Q_reg;
__constant__ float user_bias_reg;
__constant__ float item_bias_reg;
__constant__ bool is_train;
} // namespace config

__device__ float get_prediction(int factors, const float *p, const float *q, float user_bias, float item_bias,
                                float global_bias) {
    float pred = global_bias + user_bias + item_bias;
    for (int f = 0; f < factors; ++f) {
        pred += q[f] * p[f];
    }
    return pred;
}

// First pass: deterministically select a unique winner (min user id) per item
__global__ void select_item_owner(const int *indptr, const int *indices, const int *random_offsets, int n_rows,
                                  int *item_owner) {
    int user = blockDim.x * blockIdx.x + threadIdx.x;
    if (user >= n_rows)
        return;

    int low = indptr[user];
    int high = indptr[user + 1];
    if (low == high)
        return;

    int width = high - low;
    int choice = random_offsets[user] % width;
    int y_i = low + choice;
    int item = indices[y_i];

    // Pick the smallest user id as the deterministic winner per item
    atomicMin(&item_owner[item], user);
}

// Second pass: apply updates; only the selected owner updates Q/item_bias
__global__ void sgd_update_deterministic(const int *indptr, const int *indices, const float *data, float *P,
                                         const float *Q, float *Q_target, int n_rows, float *user_bias,
                                         const float *item_bias, float *item_bias_target, const int *random_offsets,
                                         float global_bias, const int *item_owner, unsigned char *item_is_updated) {
    int user = blockDim.x * blockIdx.x + threadIdx.x;
    if (user >= n_rows)
        return;

    int low = indptr[user];
    int high = indptr[user + 1];
    if (low == high)
        return;

    int width = high - low;
    int choice = random_offsets[user] % width;
    int y_i = low + choice;

    int item = indices[y_i];
    float ub = user_bias[user];
    float ib = item_bias[item];

    float error = data[y_i] - get_prediction(config::n_factors, &P[user * config::n_factors],
                                             &Q[item * config::n_factors], ub, ib, global_bias);

    // Update P (per-user, no conflicts with one thread per user)
    for (int f = 0; f < config::n_factors; ++f) {
        float P_old = P[user * config::n_factors + f];
        float Q_old = Q[item * config::n_factors + f];
        P[user * config::n_factors + f] = P_old + config::learning_rate * (error * Q_old - config::P_reg * P_old);
    }

    // Update user bias
    user_bias[user] += config::learning_rate * (error - config::user_bias_reg * ub);

    // Only the selected owner updates item parameters (deterministic winner)
    if (config::is_train && item_owner[item] == user) {
        for (int f = 0; f < config::n_factors; ++f) {
            float P_old = P[user * config::n_factors + f];
            float Q_old = Q[item * config::n_factors + f];
            Q_target[item * config::n_factors + f] =
                Q_old + config::learning_rate * (error * P_old - config::Q_reg * Q_old);
        }
        item_bias_target[item] = ib + config::learning_rate * (error - config::item_bias_reg * ib);
        item_is_updated[item] = 1u;
    }
}

int main(int argc, char **argv) {
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " <num_users> <num_items> <items_per_user>\n";
        return EXIT_FAILURE;
    }

    int num_users = std::stoi(argv[1]);
    int num_items = std::stoi(argv[2]);
    int items_per_user = std::stoi(argv[3]);

    if (num_users <= 0 || num_items <= 0 || items_per_user <= 0 || items_per_user > num_items) {
        std::cerr << "Arguments must satisfy num_users>0, num_items>0, "
                  << "0<items_per_user<=num_items.\n";
        return EXIT_FAILURE;
    }

    const int latent_factors = 8;
    const float learning_rate = 0.01f;
    const float lambda_p = 0.02f;
    const float lambda_q = 0.02f;
    const float lambda_user_bias = 0.02f;
    const float lambda_item_bias = 0.02f;
    const bool is_train = true;

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::n_factors), &latent_factors, sizeof(int)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::learning_rate), &learning_rate, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::P_reg), &lambda_p, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::Q_reg), &lambda_q, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::user_bias_reg), &lambda_user_bias, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::item_bias_reg), &lambda_item_bias, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::is_train), &is_train, sizeof(bool)));

    int nnz = num_users * items_per_user;

    std::mt19937 rng(HOST_RANDOM_SEED);
    std::uniform_real_distribution<float> rating_dist(1.0f, 5.0f);
    std::uniform_real_distribution<float> factor_dist(-0.5f, 0.5f);
    std::uniform_int_distribution<int> item_dist(0, num_items - 1);
    std::uniform_int_distribution<int> choice_dist(0, items_per_user - 1);

    std::vector<int> h_indptr(num_users + 1);
    for (int u = 0; u <= num_users; ++u) {
        h_indptr[u] = u * items_per_user;
    }

    std::vector<int> h_indices(nnz);
    std::vector<float> h_data(nnz);
    for (int idx = 0; idx < nnz; ++idx) {
        h_indices[idx] = item_dist(rng);
        h_data[idx] = rating_dist(rng);
    }

    std::vector<float> h_P(num_users * latent_factors);
    std::vector<float> h_Q(num_items * latent_factors);
    std::vector<float> h_Q_target(num_items * latent_factors);
    for (float &v : h_P) {
        v = factor_dist(rng);
    }
    for (int i = 0; i < num_items * latent_factors; ++i) {
        float val = factor_dist(rng);
        h_Q[i] = val;
        h_Q_target[i] = val;
    }

    std::vector<float> h_user_bias(num_users);
    std::vector<float> h_item_bias(num_items);
    std::vector<float> h_item_bias_target(num_items);
    for (int u = 0; u < num_users; ++u) {
        h_user_bias[u] = factor_dist(rng);
    }
    for (int i = 0; i < num_items; ++i) {
        float val = factor_dist(rng);
        h_item_bias[i] = val;
        h_item_bias_target[i] = val;
    }

    std::vector<unsigned char> h_item_updated(num_items, 0u);
    std::vector<int> h_random_choice(num_users);
    for (int u = 0; u < num_users; ++u) {
        h_random_choice[u] = choice_dist(rng);
    }

    float global_bias = factor_dist(rng);

    int *d_indptr = nullptr;
    int *d_indices = nullptr;
    float *d_data = nullptr;
    float *d_P = nullptr;
    float *d_Q = nullptr;
    float *d_Q_target = nullptr;
    float *d_user_bias = nullptr;
    float *d_item_bias = nullptr;
    float *d_item_bias_target = nullptr;
    unsigned char *d_item_is_updated = nullptr;
    int *d_random_choice = nullptr;
    int *d_item_owner = nullptr;

    CUDA_CHECK(hipMalloc(&d_indptr, h_indptr.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_indices, h_indices.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_data, h_data.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_P, h_P.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_Q, h_Q.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_Q_target, h_Q_target.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_user_bias, h_user_bias.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_item_bias, h_item_bias.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_item_bias_target, h_item_bias_target.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_item_is_updated, h_item_updated.size() * sizeof(unsigned char)));
    CUDA_CHECK(hipMalloc(&d_random_choice, h_random_choice.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_item_owner, num_items * sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_indptr, h_indptr.data(), h_indptr.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_indices, h_indices.data(), h_indices.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_data, h_data.data(), h_data.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_P, h_P.data(), h_P.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Q, h_Q.data(), h_Q.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Q_target, h_Q_target.data(), h_Q_target.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_user_bias, h_user_bias.data(), h_user_bias.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_item_bias, h_item_bias.data(), h_item_bias.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_item_bias_target, h_item_bias_target.data(), h_item_bias_target.size() * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_item_is_updated, h_item_updated.data(), h_item_updated.size() * sizeof(unsigned char),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_random_choice, h_random_choice.data(), h_random_choice.size() * sizeof(int),
                          hipMemcpyHostToDevice));
    // Initialize item_owner to a large value so atomicMin selects actual user ids
    {
        std::vector<int> h_item_owner(num_items, std::numeric_limits<int>::max());
        CUDA_CHECK(
            hipMemcpy(d_item_owner, h_item_owner.data(), h_item_owner.size() * sizeof(int), hipMemcpyHostToDevice));
    }

    dim3 block_dim(std::min(128, num_users));
    if (block_dim.x == 0)
        block_dim.x = 1;
    dim3 grid_dim((num_users + block_dim.x - 1) / block_dim.x);
    if (grid_dim.x == 0) {
        grid_dim.x = 1;
    }

    // First pass: pick deterministic item owner per item
    select_item_owner<<<grid_dim, block_dim>>>(d_indptr, d_indices, d_random_choice, num_users, d_item_owner);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Second pass: apply updates; winners update item state
    sgd_update_deterministic<<<grid_dim, block_dim>>>(d_indptr, d_indices, d_data, d_P, d_Q, d_Q_target, num_users,
                                                      d_user_bias, d_item_bias, d_item_bias_target, d_random_choice,
                                                      global_bias, d_item_owner, d_item_is_updated);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_P.data(), d_P, h_P.size() * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_Q_target.data(), d_Q_target, h_Q_target.size() * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_user_bias.data(), d_user_bias, h_user_bias.size() * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_item_bias_target.data(), d_item_bias_target, h_item_bias_target.size() * sizeof(float),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_item_updated.data(), d_item_is_updated, h_item_updated.size() * sizeof(unsigned char),
                          hipMemcpyDeviceToHost));

    std::cout << std::fixed << std::setprecision(6);
    for (float v : h_P) {
        std::cout << v << ' ';
    }
    for (float v : h_Q_target) {
        std::cout << v << ' ';
    }
    for (float v : h_user_bias) {
        std::cout << v << ' ';
    }
    for (float v : h_item_bias_target) {
        std::cout << v << ' ';
    }
    for (unsigned char v : h_item_updated) {
        std::cout << static_cast<int>(v) << ' ';
    }
    std::cout << '\n';

    hipFree(d_item_owner);
    hipFree(d_random_choice);
    hipFree(d_item_is_updated);
    hipFree(d_item_bias_target);
    hipFree(d_item_bias);
    hipFree(d_user_bias);
    hipFree(d_Q_target);
    hipFree(d_Q);
    hipFree(d_P);
    hipFree(d_data);
    hipFree(d_indices);
    hipFree(d_indptr);

    return EXIT_SUCCESS;
}
