#include <hip/hip_runtime.h>

#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#include <limits>
#include <random>
#include <vector>

#define SEED 24680

#define CUDA_CHECK(call)                                                                                               \
    do {                                                                                                               \
        hipError_t err__ = (call);                                                                                    \
        if (err__ != hipSuccess) {                                                                                    \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " code=" << err__ << " ("                  \
                      << hipGetErrorString(err__) << ")\n";                                                           \
            std::exit(EXIT_FAILURE);                                                                                   \
        }                                                                                                              \
    } while (0)

namespace config {
__constant__ int n_factors;
__constant__ float learning_rate;
__constant__ float P_reg;
__constant__ float Q_reg;
__constant__ float user_bias_reg;
__constant__ float item_bias_reg;
__constant__ bool is_train;
} // namespace config

__device__ float get_prediction(int factors, const float *p, const float *q, float user_bias, float item_bias,
                                float global_bias) {
    float pred = global_bias + user_bias + item_bias;
    for (int f = 0; f < factors; ++f)
        pred += q[f] * p[f];
    return pred;
}

__global__ void select_item_owner(const int *indptr, const int *indices, const int *random_offsets, int n_rows,
                                  int *item_owner) {
    int user = blockDim.x * blockIdx.x + threadIdx.x;
    if (user >= n_rows)
        return;
    int low = indptr[user], high = indptr[user + 1];
    if (low == high)
        return;
    int width = high - low;
    int choice = random_offsets[user] % width;
    int y_i = low + choice;
    int item = indices[y_i];
    atomicMin(&item_owner[item], user);
}

__global__ void sgd_update_deterministic(const int *indptr, const int *indices, const float *data, float *P,
                                         const float *Q, float *Q_target, int n_rows, float *user_bias,
                                         const float *item_bias, float *item_bias_target, const int *random_offsets,
                                         float global_bias, const int *item_owner, unsigned char *item_is_updated) {
    int user = blockDim.x * blockIdx.x + threadIdx.x;
    if (user >= n_rows)
        return;
    int low = indptr[user], high = indptr[user + 1];
    if (low == high)
        return;
    int width = high - low;
    int choice = random_offsets[user] % width;
    int y_i = low + choice;
    int item = indices[y_i];
    float ub = user_bias[user];
    float ib = item_bias[item];
    float error = data[y_i] - get_prediction(config::n_factors, &P[user * config::n_factors],
                                             &Q[item * config::n_factors], ub, ib, global_bias);
    for (int f = 0; f < config::n_factors; ++f) {
        float P_old = P[user * config::n_factors + f];
        float Q_old = Q[item * config::n_factors + f];
        P[user * config::n_factors + f] = P_old + config::learning_rate * (error * Q_old - config::P_reg * P_old);
    }
    user_bias[user] += config::learning_rate * (error - config::user_bias_reg * ub);
    if (config::is_train && item_owner[item] == user) {
        for (int f = 0; f < config::n_factors; ++f) {
            float P_old = P[user * config::n_factors + f];
            float Q_old = Q[item * config::n_factors + f];
            Q_target[item * config::n_factors + f] =
                Q_old + config::learning_rate * (error * P_old - config::Q_reg * Q_old);
        }
        item_bias_target[item] = ib + config::learning_rate * (error - config::item_bias_reg * ib);
        item_is_updated[item] = 1u;
    }
}

static inline float rand_uniform_float32(std::mt19937 &rng) {
    return static_cast<float>(rng()); // 范围 [0, 4.294967295e9]，但精度不足
}

int main(int argc, char **argv) {
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " <num_users> <num_items> <items_per_user>\n";
        return EXIT_FAILURE;
    }
    int num_users = std::stoi(argv[1]);
    int num_items = std::stoi(argv[2]);
    int items_per_user = std::stoi(argv[3]);
    if (num_users <= 0 || num_items <= 0 || items_per_user <= 0 || items_per_user > num_items) {
        std::cerr << "Arguments must satisfy num_users>0, num_items>0, 0<items_per_user<=num_items.\n";
        return EXIT_FAILURE;
    }

    srand(SEED);
    std::mt19937 rng(SEED);

    const int latent_factors = 8;
    const float learning_rate = 0.01f;
    const float lambda_p = 0.02f;
    const float lambda_q = 0.02f;
    const float lambda_user_bias = 0.02f;
    const float lambda_item_bias = 0.02f;
    const bool is_train = true;
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::n_factors), &latent_factors, sizeof(int)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::learning_rate), &learning_rate, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::P_reg), &lambda_p, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::Q_reg), &lambda_q, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::user_bias_reg), &lambda_user_bias, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::item_bias_reg), &lambda_item_bias, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(config::is_train), &is_train, sizeof(bool)));

    int nnz = num_users * items_per_user;
    std::vector<int> h_indptr(num_users + 1);
    for (int u = 0; u <= num_users; ++u)
        h_indptr[u] = u * items_per_user;
    std::vector<int> h_indices(nnz);
    std::vector<float> h_data(nnz);
    for (int i = 0; i < nnz; ++i) {
        h_indices[i] = rand() % num_items;
        h_data[i] = (rand() % 2 == 0) ? std::numeric_limits<float>::quiet_NaN() : rand_uniform_float32(rng);
    }
    std::vector<float> h_P(num_users * latent_factors);
    std::vector<float> h_Q(num_items * latent_factors);
    std::vector<float> h_Q_target(num_items * latent_factors);
    for (size_t i = 0; i < h_P.size(); ++i)
        h_P[i] = (rand() % 2 == 0) ? std::numeric_limits<float>::quiet_NaN() : rand_uniform_float32(rng);
    for (size_t i = 0; i < h_Q.size(); ++i)
        h_Q[i] = (rand() % 2 == 0) ? std::numeric_limits<float>::quiet_NaN() : rand_uniform_float32(rng);
    h_Q_target = h_Q;
    std::vector<float> h_user_bias(num_users);
    std::vector<float> h_item_bias(num_items);
    std::vector<float> h_item_bias_target(num_items);
    for (int i = 0; i < num_users; ++i)
        h_user_bias[i] = (rand() % 2 == 0) ? std::numeric_limits<float>::quiet_NaN() : rand_uniform_float32(rng);
    for (int i = 0; i < num_items; ++i)
        h_item_bias[i] = (rand() % 2 == 0) ? std::numeric_limits<float>::quiet_NaN() : rand_uniform_float32(rng);
    h_item_bias_target = h_item_bias;

    std::vector<unsigned char> h_item_updated(num_items, 0u);
    std::vector<int> h_random_choice(num_users);
    for (int u = 0; u < num_users; ++u)
        h_random_choice[u] = rand() % items_per_user;

    float global_bias = 0.0f; // keep finite

    int *d_indptr = nullptr, *d_indices = nullptr;
    float *d_data = nullptr, *d_P = nullptr, *d_Q = nullptr, *d_Q_target = nullptr, *d_user_bias = nullptr,
          *d_item_bias = nullptr, *d_item_bias_target = nullptr;
    unsigned char *d_item_is_updated = nullptr;
    int *d_random_choice = nullptr, *d_item_owner = nullptr;
    CUDA_CHECK(hipMalloc(&d_indptr, h_indptr.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_indices, h_indices.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_data, h_data.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_P, h_P.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_Q, h_Q.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_Q_target, h_Q_target.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_user_bias, h_user_bias.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_item_bias, h_item_bias.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_item_bias_target, h_item_bias_target.size() * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_item_is_updated, h_item_updated.size() * sizeof(unsigned char)));
    CUDA_CHECK(hipMalloc(&d_random_choice, h_random_choice.size() * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_item_owner, num_items * sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_indptr, h_indptr.data(), h_indptr.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_indices, h_indices.data(), h_indices.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_data, h_data.data(), h_data.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_P, h_P.data(), h_P.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Q, h_Q.data(), h_Q.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Q_target, h_Q_target.data(), h_Q_target.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_user_bias, h_user_bias.data(), h_user_bias.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_item_bias, h_item_bias.data(), h_item_bias.size() * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_item_bias_target, h_item_bias_target.data(), h_item_bias_target.size() * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_item_is_updated, h_item_updated.data(), h_item_updated.size() * sizeof(unsigned char),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_random_choice, h_random_choice.data(), h_random_choice.size() * sizeof(int),
                          hipMemcpyHostToDevice));
    {
        std::vector<int> tmp(num_items, std::numeric_limits<int>::max());
        CUDA_CHECK(hipMemcpy(d_item_owner, tmp.data(), tmp.size() * sizeof(int), hipMemcpyHostToDevice));
    }

    dim3 block_dim(std::min(128, num_users));
    if (block_dim.x == 0)
        block_dim.x = 1;
    dim3 grid_dim((num_users + block_dim.x - 1) / block_dim.x);
    if (grid_dim.x == 0)
        grid_dim.x = 1;
    select_item_owner<<<grid_dim, block_dim>>>(d_indptr, d_indices, d_random_choice, num_users, d_item_owner);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    sgd_update_deterministic<<<grid_dim, block_dim>>>(d_indptr, d_indices, d_data, d_P, d_Q, d_Q_target, num_users,
                                                      d_user_bias, d_item_bias, d_item_bias_target, d_random_choice,
                                                      global_bias, d_item_owner, d_item_is_updated);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_P.data(), d_P, h_P.size() * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_Q_target.data(), d_Q_target, h_Q_target.size() * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_user_bias.data(), d_user_bias, h_user_bias.size() * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_item_bias_target.data(), d_item_bias_target, h_item_bias_target.size() * sizeof(float),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_item_updated.data(), d_item_is_updated, h_item_updated.size() * sizeof(unsigned char),
                          hipMemcpyDeviceToHost));

    std::cout << std::fixed << std::setprecision(6);
    for (float v : h_P)
        std::cout << v << ' ';
    for (float v : h_Q_target)
        std::cout << v << ' ';
    for (float v : h_user_bias)
        std::cout << v << ' ';
    for (float v : h_item_bias_target)
        std::cout << v << ' ';
    for (unsigned char v : h_item_updated)
        std::cout << (int)v << ' ';
    std::cout << '\n';

    hipFree(d_item_owner);
    hipFree(d_random_choice);
    hipFree(d_item_is_updated);
    hipFree(d_item_bias_target);
    hipFree(d_item_bias);
    hipFree(d_user_bias);
    hipFree(d_Q_target);
    hipFree(d_Q);
    hipFree(d_P);
    hipFree(d_data);
    hipFree(d_indices);
    hipFree(d_indptr);
    return 0;
}
