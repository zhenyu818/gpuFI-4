#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define M_SEED 5311
#define BLOCK_SIZE 256

// kernel1: dot product
__global__ void kernel1(const __half *__restrict__ key, const __half *__restrict__ query,
                        __half *__restrict__ dot_product,
                        float *__restrict__ exp_sum, // float 累加
                        const int n, const int d) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float sum = 0.0f;
        for (int j = 0; j < d; j++) {
            sum += __half2float(key[i * d + j]) * __half2float(query[j]);
        }
        dot_product[i] = __float2half(sum);
        atomicAdd(exp_sum, expf(sum));
    }
}

// kernel2: softmax
__global__ void kernel2(const float *__restrict__ exp_sum, const __half *__restrict__ dot_product,
                        __half *__restrict__ score, const int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float dp = __half2float(dot_product[i]);
        float s = expf(dp) / exp_sum[0];
        score[i] = __float2half(s);
    }
}

// kernel3: weighted sum
__global__ void kernel3(const __half *__restrict__ score, const __half *__restrict__ value, __half *__restrict__ output,
                        const int n, const int d) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < d) {
        float sum = 0.0f;
        for (int i = 0; i < n; i++) {
            sum += __half2float(score[i]) * __half2float(value[i * d + j]);
        }
        output[j] = __float2half(sum);
    }
}

// device function
__half *attention_device(const __half *key, const __half *value, const __half *query, const int n, const int d,
                         const int repeat) {
    __half *d_key;
    hipMalloc((void **)&d_key, n * d * sizeof(__half));
    hipMemcpy(d_key, key, n * d * sizeof(__half), hipMemcpyHostToDevice);

    __half *d_value;
    hipMalloc((void **)&d_value, n * d * sizeof(__half));
    hipMemcpy(d_value, value, n * d * sizeof(__half), hipMemcpyHostToDevice);

    __half *d_query;
    hipMalloc((void **)&d_query, d * sizeof(__half));
    hipMemcpy(d_query, query, d * sizeof(__half), hipMemcpyHostToDevice);

    __half *d_dot_product;
    hipMalloc((void **)&d_dot_product, n * sizeof(__half));

    __half *d_score;
    hipMalloc((void **)&d_score, n * sizeof(__half));

    float *d_exp_sum;
    hipMalloc((void **)&d_exp_sum, sizeof(float));

    __half *output = (__half *)malloc(d * sizeof(__half));
    __half *d_output;
    hipMalloc((void **)&d_output, d * sizeof(__half));

    dim3 n_grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 n_block(BLOCK_SIZE);
    dim3 d_grid((d + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 d_block(BLOCK_SIZE);

    hipDeviceSynchronize();

    for (int k = 0; k < repeat; k++) {
        hipMemset(d_exp_sum, 0, sizeof(float));

        kernel1<<<n_grid, n_block>>>(d_key, d_query, d_dot_product, d_exp_sum, n, d);
        kernel2<<<n_grid, n_block>>>(d_exp_sum, d_dot_product, d_score, n);
        kernel3<<<d_grid, d_block>>>(d_score, d_value, d_output, n, d);
    }

    hipDeviceSynchronize();

    hipMemcpy(output, d_output, d * sizeof(__half), hipMemcpyDeviceToHost);
    hipFree(d_score);
    hipFree(d_value);
    hipFree(d_output);
    hipFree(d_key);
    hipFree(d_dot_product);
    hipFree(d_exp_sum);
    return output;
}

// ---- 修改后的输入生成函数：包含 NaN ----
static void generate_input_with_nan(__half *buf, int total_len) {
    srand(M_SEED);
    for (int i = 0; i < total_len; i++) {
        if (rand() % 2 == 0) {
            buf[i] = __float2half(NAN); // 50% 概率 NaN
        } else {
            // 生成 -1.0f 到 1.0f 范围的随机浮点数
            float r = (float)rand() / (float)RAND_MAX; // [0,1]
            float val = r * 2.0f - 1.0f;               // 映射到 [-1,1]
            buf[i] = __float2half(val);
        }
    }
}
static void generate_input_with_nan_2_2(__half *buf, int total_len) {
    srand(M_SEED);
    for (int i = 0; i < total_len; i++) {
        if (rand() % 2 == 0) {
            buf[i] = __float2half(NAN); // 50% 概率 NaN
        } else {
            // 生成 -2.0f 到 2.0f 范围的随机浮点数
            float r = (float)rand() / (float)RAND_MAX; // [0,1]
            float val = r * 4.0f - 2.0f;               // 映射到 [-2,2]
            buf[i] = __float2half(val);
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s <rows> <columns>\n", argv[0]);
        return 1;
    }
    const int n = atoi(argv[1]);
    const int d = atoi(argv[2]);
    const int r = 1;

    __half *key = (__half *)malloc(n * d * sizeof(__half));
    __half *value = (__half *)malloc(n * d * sizeof(__half));
    __half *query = (__half *)malloc(d * sizeof(__half));

    generate_input_with_nan(key, n * d);
    generate_input_with_nan(value, n * d);
    generate_input_with_nan_2_2(query, d);

    __half *dout = attention_device(key, value, query, n, d, r);

    for (int i = 0; i < d; i++)
        printf("%.3f ", __half2float(dout[i]));
    printf("\n");
    free(key);
    free(value);
    free(query);
    free(dout);
    return 0;
}
