/* Modified variant: different random seed */
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAXDISTANCE (200)
#define RANDOM_SEED 3
#define BLOCK_SIZE 16

__global__ void floydWarshallPass(unsigned int *__restrict__ pathDistanceBuffer, unsigned int *__restrict__ pathBuffer,
                                  const unsigned int numNodes, const unsigned int pass) {
    int xValue = threadIdx.x + blockIdx.x * blockDim.x;
    int yValue = threadIdx.y + blockIdx.y * blockDim.y;
    int k = pass;
    int oldWeight = pathDistanceBuffer[yValue * numNodes + xValue];
    int tempWeight = pathDistanceBuffer[yValue * numNodes + k] + pathDistanceBuffer[k * numNodes + xValue];
    if (tempWeight < oldWeight) {
        pathDistanceBuffer[yValue * numNodes + xValue] = tempWeight;
        pathBuffer[yValue * numNodes + xValue] = k;
    }
}

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Usage: %s <number of nodes>\n", argv[0]);
        return 1;
    }
    unsigned int numNodes = atoi(argv[1]);
    unsigned int matrixSizeBytes = numNodes * numNodes * sizeof(unsigned int);
    unsigned int *pathDistanceMatrix = (unsigned int *)malloc(matrixSizeBytes);
    assert(pathDistanceMatrix);
    unsigned int *pathMatrix = (unsigned int *)malloc(matrixSizeBytes);
    assert(pathMatrix);
    srand(RANDOM_SEED);
    for (unsigned int i = 0; i < numNodes; i++) {
        for (unsigned int j = 0; j < numNodes; j++) {
            int index = i * numNodes + j;
            pathDistanceMatrix[index] = rand() % (MAXDISTANCE + 1);
        }
    }
    for (unsigned int i = 0; i < numNodes; ++i) {
        pathDistanceMatrix[i * numNodes + i] = 0;
    }
    for (unsigned int i = 0; i < numNodes; ++i) {
        for (unsigned int j = 0; j < i; ++j) {
            pathMatrix[i * numNodes + j] = i;
            pathMatrix[j * numNodes + i] = j;
        }
        pathMatrix[i * numNodes + i] = i;
    }
    dim3 grids(numNodes / BLOCK_SIZE, numNodes / BLOCK_SIZE);
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    unsigned int *pathDistanceBuffer, *pathBuffer;
    hipMalloc((void **)&pathDistanceBuffer, matrixSizeBytes);
    hipMalloc((void **)&pathBuffer, matrixSizeBytes);
    hipMemcpy(pathDistanceBuffer, pathDistanceMatrix, matrixSizeBytes, hipMemcpyHostToDevice);
    hipMemcpy(pathBuffer, pathMatrix, matrixSizeBytes, hipMemcpyHostToDevice);
    floydWarshallPass<<<grids, threads>>>(pathDistanceBuffer, pathBuffer, numNodes, 0);
    hipDeviceSynchronize();
    hipMemcpy(pathDistanceMatrix, pathDistanceBuffer, matrixSizeBytes, hipMemcpyDeviceToHost);
    hipMemcpy(pathMatrix, pathBuffer, matrixSizeBytes, hipMemcpyDeviceToHost);
    for (unsigned int i = 0; i < numNodes * numNodes; i++) {
        printf("%u", pathDistanceMatrix[i]);
        if (i < numNodes * numNodes - 1)
            printf(" ");
    }
    printf("\n");
    for (unsigned int i = 0; i < numNodes * numNodes; i++) {
        printf("%u", pathMatrix[i]);
        if (i < numNodes * numNodes - 1)
            printf(" ");
    }
    printf("\n");
    hipFree(pathDistanceBuffer);
    hipFree(pathBuffer);
    free(pathDistanceMatrix);
    free(pathMatrix);
    return 0;
}
