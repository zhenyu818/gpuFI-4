/* Inputs all zeros */
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

__global__ void floydWarshallPass(unsigned int *__restrict__ pathDistanceBuffer, unsigned int *__restrict__ pathBuffer,
                                  const unsigned int numNodes, const unsigned int pass) {
    int xValue = threadIdx.x + blockIdx.x * blockDim.x;
    int yValue = threadIdx.y + blockIdx.y * blockDim.y;
    int k = pass;
    int oldWeight = pathDistanceBuffer[yValue * numNodes + xValue];
    int tempWeight = pathDistanceBuffer[yValue * numNodes + k] + pathDistanceBuffer[k * numNodes + xValue];
    if (tempWeight < oldWeight) {
        pathDistanceBuffer[yValue * numNodes + xValue] = tempWeight;
        pathBuffer[yValue * numNodes + xValue] = k;
    }
}

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Usage: %s <number of nodes>\n", argv[0]);
        return 1;
    }
    unsigned int numNodes = atoi(argv[1]);
    unsigned int matrixSizeBytes = numNodes * numNodes * sizeof(unsigned int);
    unsigned int *pathDistanceMatrix = (unsigned int *)malloc(matrixSizeBytes);
    assert(pathDistanceMatrix);
    unsigned int *pathMatrix = (unsigned int *)malloc(matrixSizeBytes);
    assert(pathMatrix);
    for (unsigned int i = 0; i < numNodes; i++) {
        for (unsigned int j = 0; j < numNodes; j++) {
            int index = i * numNodes + j;
            pathDistanceMatrix[index] = 0;
        }
    }
    for (unsigned int i = 0; i < numNodes; ++i) {
        pathDistanceMatrix[i * numNodes + i] = 0;
    }
    for (unsigned int i = 0; i < numNodes; ++i) {
        for (unsigned int j = 0; j < i; ++j) {
            pathMatrix[i * numNodes + j] = i;
            pathMatrix[j * numNodes + i] = j;
        }
        pathMatrix[i * numNodes + i] = i;
    }
    dim3 grids(numNodes / BLOCK_SIZE, numNodes / BLOCK_SIZE);
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    unsigned int *pathDistanceBuffer, *pathBuffer;
    hipMalloc((void **)&pathDistanceBuffer, matrixSizeBytes);
    hipMalloc((void **)&pathBuffer, matrixSizeBytes);
    hipMemcpy(pathDistanceBuffer, pathDistanceMatrix, matrixSizeBytes, hipMemcpyHostToDevice);
    hipMemcpy(pathBuffer, pathMatrix, matrixSizeBytes, hipMemcpyHostToDevice);
    floydWarshallPass<<<grids, threads>>>(pathDistanceBuffer, pathBuffer, numNodes, 0);
    hipDeviceSynchronize();
    hipMemcpy(pathDistanceMatrix, pathDistanceBuffer, matrixSizeBytes, hipMemcpyDeviceToHost);
    hipMemcpy(pathMatrix, pathBuffer, matrixSizeBytes, hipMemcpyDeviceToHost);
    FILE *fp = fopen("result.txt", "r");
    if (!fp) {
        printf("Failed to open result.txt\n");
        return 1;
    }
    unsigned int *expected_distance = (unsigned int *)malloc(matrixSizeBytes);
    unsigned int *expected_path = (unsigned int *)malloc(matrixSizeBytes);
    for (unsigned int i = 0; i < numNodes * numNodes; i++) {
        if (fscanf(fp, "%u", &expected_distance[i]) != 1) {
            printf("Failed to read expected distance data\n");
            fclose(fp);
            free(expected_distance);
            free(expected_path);
            return 1;
        }
    }
    for (unsigned int i = 0; i < numNodes * numNodes; i++) {
        if (fscanf(fp, "%u", &expected_path[i]) != 1) {
            printf("Failed to read expected path data\n");
            fclose(fp);
            free(expected_distance);
            free(expected_path);
            return 1;
        }
    }
    fclose(fp);
    bool match_distance = true;
    for (unsigned int i = 0; i < numNodes * numNodes; i++) {
        if (pathDistanceMatrix[i] != expected_distance[i]) {
            match_distance = false;
            break;
        }
    }
    bool match_path = true;
    for (unsigned int i = 0; i < numNodes * numNodes; i++) {
        if (pathMatrix[i] != expected_path[i]) {
            match_path = false;
            break;
        }
    }
    printf((match_distance && match_path) ? "Fault Injection Test Success!\n" : "Fault Injection Test Failed!\n");
    free(expected_distance);
    free(expected_path);
    hipFree(pathDistanceBuffer);
    hipFree(pathBuffer);
    free(pathDistanceMatrix);
    free(pathMatrix);
    return 0;
}
