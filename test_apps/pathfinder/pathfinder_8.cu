
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <string.h>

struct timeval tv;
struct timeval tv_total_start, tv_total_end;
struct timeval tv_h2d_start, tv_h2d_end;
struct timeval tv_d2h_start, tv_d2h_end;
struct timeval tv_kernel_start, tv_kernel_end;
struct timeval tv_mem_alloc_start, tv_mem_alloc_end;
struct timeval tv_close_start, tv_close_end;
float init_time = 0, mem_alloc_time = 0, h2d_time = 0, kernel_time = 0,
      d2h_time = 0, close_time = 0, total_time = 0;

#define BLOCK_SIZE 256
#define STR_SIZE 256
#define DEVICE 0
#define HALO 1 // halo width along one direction when advancing to the next iteration

#define M_SEED 3415
#define EXP_NAME "1-1"

//#define BENCH_PRINT

void run(int argc, char** argv);

int rows, cols;
int* data;
int** wall;
int* result;
int pyramid_height;

// 从pathfinder_gen_input_1.cu集成的输入生成函数
static void generate_input_1(int argc, char **argv)
{
	if (argc == 4) {
		cols = atoi(argv[1]);
		rows = atoi(argv[2]);
		pyramid_height = atoi(argv[3]);
	} else {
		printf("Usage: dynproc row_len col_len pyramid_height\n");
		exit(0);
	}

	data = new int[rows*cols];
	wall = new int*[rows];
	for(int n=0; n<rows; n++)
		wall[n]=data+cols*n;
	result = new int[cols];

	// 生成包含对抗性模式的输入数据
	srand(M_SEED);
	
	// 对抗性模式1: 极值模式 - 在关键位置放置极大值和极小值
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			// 在边界和关键路径上放置极值
			if (i == 0 || i == rows-1 || j == 0 || j == cols-1) {
				// 边界放置极大值，增加边界处理复杂度
				wall[i][j] = (i + j) % 2 == 0 ? 999 : 1;
			} else if (i % 3 == 0 && j % 3 == 0) {
				// 在3的倍数位置放置极小值，创建"陷阱"
				wall[i][j] = 0;
			} else if (i == rows/2 && j == cols/2) {
				// 中心位置放置极大值，增加中心路径计算复杂度
				wall[i][j] = 999;
			} else if ((i + j) % 7 == 0) {
				// 在特定模式位置放置交替的极值
				wall[i][j] = (i + j) % 14 == 0 ? 999 : 1;
			} else {
				// 其他位置生成中等范围的随机值，但偏向于创建复杂路径
				int base_val = rand() % 20;
				// 增加一些"噪声"，使路径规划更复杂
				if (base_val < 5) {
					wall[i][j] = rand() % 100 + 50;  // 50-149的高值
				} else if (base_val < 10) {
					wall[i][j] = rand() % 10;        // 0-9的低值
				} else {
					wall[i][j] = rand() % 30 + 10;   // 10-39的中等值
				}
			}
		}
	}
	
	// 对抗性模式2: 创建"迷宫"模式 - 在特定行创建高成本路径
	for (int i = 1; i < rows-1; i += 4) {
		for (int j = 0; j < cols; j++) {
			if (j % 2 == 0) {
				wall[i][j] = 888;  // 创建高成本行
			}
		}
	}
	
	// 对抗性模式3: 在关键计算路径上放置交替的高低值
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			if ((i + j) % 5 == 0 && i > 0 && i < rows-1 && j > 0 && j < cols-1) {
				// 在内部位置创建交替模式，增加动态规划的计算复杂度
				wall[i][j] = (i + j) % 10 == 0 ? 777 : 3;
			}
		}
	}
	
	// 对抗性模式4: 在算法可能优化的路径上放置挑战性值
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			if (i == j || i == cols-1-j) {
				// 对角线位置放置特殊值，挑战算法的优化策略
				wall[i][j] = (i + j) % 3 == 0 ? 666 : 5;
			}
		}
	}
	
    printf("Adversarial input generated with seed %d\n", M_SEED);
    printf("Input matrix (%dx%d):\n", rows, cols);
    for(int i=0; i<rows; i++) {
        for(int j=0; j<cols; j++) {
            printf("%3d ", wall[i][j]);
        }
        printf("\n");
    }
}

void
init(int argc, char** argv)
{
	// 调用集成的输入生成函数
	generate_input_1(argc, argv);
}

void 
fatal(char *s)
{
	fprintf(stderr, "error: %s\n", s);

}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__global__ void dynproc_kernel(
                int iteration, 
                int *gpuWall,
                int *gpuSrc,
                int *gpuResults,
                int cols, 
                int rows,
                int startStep,
                int border)
{

        __shared__ int prev[BLOCK_SIZE];
        __shared__ int result[BLOCK_SIZE];

	int bx = blockIdx.x;
	int tx=threadIdx.x;
	
        // each block finally computes result for a small block
        // after N iterations. 
        // it is the non-overlapping small blocks that cover 
        // all the input data

        // calculate the small block size
	int small_block_cols = BLOCK_SIZE-iteration*HALO*2;

        // calculate the boundary for the block according to 
        // the boundary of its small block
        int blkX = small_block_cols*bx-border;
        int blkXmax = blkX+BLOCK_SIZE-1;

        // calculate the global thread coordination
	int xidx = blkX+tx;
       
        // effective range within this block that falls within 
        // the valid range of the input data
        // used to rule out computation outside the boundary.
        int validXmin = (blkX < 0) ? -blkX : 0;
        int validXmax = (blkXmax > cols-1) ? BLOCK_SIZE-1-(blkXmax-cols+1) : BLOCK_SIZE-1;

        int W = tx-1;
        int E = tx+1;
        
        W = (W < validXmin) ? validXmin : W;
        E = (E > validXmax) ? validXmax : E;

        bool isValid = IN_RANGE(tx, validXmin, validXmax);

	if(IN_RANGE(xidx, 0, cols-1)){
            prev[tx] = gpuSrc[xidx];
	}
	__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
        bool computed;
        for (int i=0; i<iteration ; i++){ 
            computed = false;
            if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
                  isValid){
                  computed = true;
                  int left = prev[W];
                  int up = prev[tx];
                  int right = prev[E];
                  int shortest = MIN(left, up);
                  shortest = MIN(shortest, right);
                  int index = cols*(startStep+i)+xidx;
                  result[tx] = shortest + gpuWall[index];
	
            }
            __syncthreads();
            if(i==iteration-1)
                break;
            if(computed)	 //Assign the computation range
                prev[tx]= result[tx];
	    __syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
      }

      // update the global memory
      // after the last iteration, only threads coordinated within the 
      // small block perform the calculation and switch on ``computed''
      if (computed){
          gpuResults[xidx]=result[tx];		
      }
}

/*
   compute N time steps
*/
int calc_path(int *gpuWall, int *gpuResult[2], int rows, int cols, \
	 int pyramid_height, int blockCols, int borderCols)
{
        dim3 dimBlock(BLOCK_SIZE);
        dim3 dimGrid(blockCols);  
	
        int src = 1, dst = 0;
	for (int t = 0; t < rows-1; t+=pyramid_height) {
            int temp = src;
            src = dst;
            dst = temp;
            dynproc_kernel<<<dimGrid, dimBlock>>>(
                MIN(pyramid_height, rows-t-1), 
                gpuWall, gpuResult[src], gpuResult[dst],
                cols,rows, t, borderCols);

            // for the measurement fairness
            hipDeviceSynchronize();
	}
        return dst;
}

int main(int argc, char** argv)
{
    int num_devices;
    hipGetDeviceCount(&num_devices);
    if (num_devices > 1) hipSetDevice(DEVICE);

    run(argc,argv);

    return EXIT_SUCCESS;
}

void run(int argc, char** argv)
{
    init(argc, argv);

    /* --------------- pyramid parameters --------------- */
    int borderCols = (pyramid_height)*HALO;
    int smallBlockCol = BLOCK_SIZE-(pyramid_height)*HALO*2;
    int blockCols = cols/smallBlockCol+((cols%smallBlockCol==0)?0:1);
	
    int *gpuWall, *gpuResult[2];
    int size = rows*cols;

    hipMalloc((void**)&gpuResult[0], sizeof(int)*cols);
    hipMalloc((void**)&gpuResult[1], sizeof(int)*cols);
    hipMemcpy(gpuResult[0], data, sizeof(int)*cols, hipMemcpyHostToDevice);
    hipMalloc((void**)&gpuWall, sizeof(int)*(size-cols));
    hipMemcpy(gpuWall, data+cols, sizeof(int)*(size-cols), hipMemcpyHostToDevice);

#ifdef  TIMING
    gettimeofday(&tv_kernel_start, NULL);
#endif

    int final_ret = calc_path(gpuWall, gpuResult, rows, cols, \
	 pyramid_height, blockCols, borderCols);

#ifdef  TIMING
    gettimeofday(&tv_kernel_end, NULL);
    tvsub(&tv_kernel_end, &tv_kernel_start, &tv);
    kernel_time += tv.tv_sec * 1000.0 + (float) tv.tv_usec / 1000.0;
#endif

    hipMemcpy(result, gpuResult[final_ret], sizeof(int)*cols, hipMemcpyDeviceToHost);
    printf("result generated\n");
    // output result array to console instead of txt file
    for (int i = 0; i < cols; ++i) {
        printf("%d%c", result[i], (i == cols - 1) ? '\n' : ' ');
    }


    hipFree(gpuWall);
    hipFree(gpuResult[0]);
    hipFree(gpuResult[1]);

    delete [] data;
    delete [] wall;
    delete [] result;
}
