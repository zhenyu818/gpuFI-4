
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>
#include <string.h>
#include <stdbool.h>

struct timeval tv;
struct timeval tv_total_start, tv_total_end;
struct timeval tv_h2d_start, tv_h2d_end;
struct timeval tv_d2h_start, tv_d2h_end;
struct timeval tv_kernel_start, tv_kernel_end;
struct timeval tv_mem_alloc_start, tv_mem_alloc_end;
struct timeval tv_close_start, tv_close_end;
float init_time = 0, mem_alloc_time = 0, h2d_time = 0, kernel_time = 0,
      d2h_time = 0, close_time = 0, total_time = 0;

#define BLOCK_SIZE 256
#define STR_SIZE 256
#define DEVICE 0
#define HALO 1 // halo width along one direction when advancing to the next iteration

#define M_SEED 3415          // 保留原随机种子
#define SPARSE_N 2           // 2:4稀疏中的N（每M个元素保留的非零值数量）
#define SPARSE_M 4           // 2:4稀疏中的M（连续元素分组大小）

//#define BENCH_PRINT

void run(int argc, char** argv);

int rows, cols;
int* data;
int** wall;
int* result;
int pyramid_height;

// 从pathfinder_gen_input_6.cu集成的2:4稀疏输入生成函数
// 生成2:4结构化稀疏的随机值（每4个连续元素中随机选2个置非零，其余置0）
static void generate_2to4_sparse_value(int *group, int group_len) {
    // 1. 初始化分组为全0（满足稀疏约束的基础）
    for (int k = 0; k < group_len; k++) {
        group[k] = 0;
    }
    
    // 2. 随机选择2个不同的位置作为非零值索引（确保每4个元素仅保留2个非零）
    bool selected[SPARSE_M] = {false};
    int selected_count = 0;
    while (selected_count < SPARSE_N) {
        int rand_idx = rand() % SPARSE_M;  // 0~3范围内随机选索引
        if (!selected[rand_idx]) {
            selected[rand_idx] = true;
            selected_count++;
        }
    }
    
    // 3. 为选中的位置生成0~9的随机非零值（匹配原代码的随机值范围）
    for (int k = 0; k < SPARSE_M; k++) {
        if (selected[k]) {
            group[k] = rand() % 10;
            // 确保非零（若随机到0则重新生成，避免与稀疏置0混淆）
            while (group[k] == 0) {
                group[k] = rand() % 10;
            }
        }
    }
}

// 生成2:4结构化稀疏的输入矩阵
static void generate_input_6(int argc, char **argv) {
    if (argc == 4) {
        cols = atoi(argv[1]);
        rows = atoi(argv[2]);
        pyramid_height = atoi(argv[3]);
        // 检查列数是否为4的整数倍（确保2:4稀疏分组完整，若不满足则自动补齐）
        if (cols % SPARSE_M != 0) {
            int new_cols = (cols / SPARSE_M + 1) * SPARSE_M;
            cols = new_cols;
        }
    } else {
        printf("Usage: dynproc row_len col_len pyramid_height\n");
        exit(0);
    }

    // 内存分配（与原代码逻辑一致）
    data = new int[rows * cols];
    wall = new int*[rows];
    for (int n = 0; n < rows; n++) {
        wall[n] = data + cols * n;
    }
    result = new int[cols];

    srand(M_SEED);  // 保留原随机种子，确保可复现性
    // 按行生成2:4稀疏数据
    for (int i = 0; i < rows; i++) {
        // 按4个元素为一组处理，确保每组满足2:4稀疏约束
        for (int j = 0; j < cols; j += SPARSE_M) {
            int group[SPARSE_M];
            generate_2to4_sparse_value(group, SPARSE_M);
            // 将稀疏分组赋值到矩阵对应位置
            for (int k = 0; k < SPARSE_M; k++) {
                wall[i][j + k] = group[k];
            }
        }
    }
}

void
init(int argc, char** argv)
{
	// 调用集成的输入生成函数
	generate_input_6(argc, argv);
}

void 
fatal(char *s)
{
	fprintf(stderr, "error: %s\n", s);

}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__global__ void dynproc_kernel(
                int iteration, 
                int *gpuWall,
                int *gpuSrc,
                int *gpuResults,
                int cols, 
                int rows,
                int startStep,
                int border)
{

        __shared__ int prev[BLOCK_SIZE];
        __shared__ int result[BLOCK_SIZE];

	int bx = blockIdx.x;
	int tx=threadIdx.x;
	
        // each block finally computes result for a small block
        // after N iterations. 
        // it is the non-overlapping small blocks that cover 
        // all the input data

        // calculate the small block size
	int small_block_cols = BLOCK_SIZE-iteration*HALO*2;

        // calculate the boundary for the block according to 
        // the boundary of its small block
        int blkX = small_block_cols*bx-border;
        int blkXmax = blkX+BLOCK_SIZE-1;

        // calculate the global thread coordination
	int xidx = blkX+tx;
       
        // effective range within this block that falls within 
        // the valid range of the input data
        // used to rule out computation outside the boundary.
        int validXmin = (blkX < 0) ? -blkX : 0;
        int validXmax = (blkXmax > cols-1) ? BLOCK_SIZE-1-(blkXmax-cols+1) : BLOCK_SIZE-1;

        int W = tx-1;
        int E = tx+1;
        
        W = (W < validXmin) ? validXmin : W;
        E = (E > validXmax) ? validXmax : E;

        bool isValid = IN_RANGE(tx, validXmin, validXmax);

	if(IN_RANGE(xidx, 0, cols-1)){
            prev[tx] = gpuSrc[xidx];
	}
	__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
        bool computed;
        for (int i=0; i<iteration ; i++){ 
            computed = false;
            if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
                  isValid){
                  computed = true;
                  int left = prev[W];
                  int up = prev[tx];
                  int right = prev[E];
                  int shortest = MIN(left, up);
                  shortest = MIN(shortest, right);
                  int index = cols*(startStep+i)+xidx;
                  result[tx] = shortest + gpuWall[index];
	
            }
            __syncthreads();
            if(i==iteration-1)
                break;
            if(computed)	 //Assign the computation range
                prev[tx]= result[tx];
	    __syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
      }

      // update the global memory
      // after the last iteration, only threads coordinated within the 
      // small block perform the calculation and switch on ``computed''
      if (computed){
          gpuResults[xidx]=result[tx];		
      }
}

/*
   compute N time steps
*/
int calc_path(int *gpuWall, int *gpuResult[2], int rows, int cols, \
	 int pyramid_height, int blockCols, int borderCols)
{
        dim3 dimBlock(BLOCK_SIZE);
        dim3 dimGrid(blockCols);  
	
        int src = 1, dst = 0;
	for (int t = 0; t < rows-1; t+=pyramid_height) {
            int temp = src;
            src = dst;
            dst = temp;
            dynproc_kernel<<<dimGrid, dimBlock>>>(
                MIN(pyramid_height, rows-t-1), 
                gpuWall, gpuResult[src], gpuResult[dst],
                cols,rows, t, borderCols);

            // for the measurement fairness
            hipDeviceSynchronize();
	}
        return dst;
}

int main(int argc, char** argv)
{
    int num_devices;
    hipGetDeviceCount(&num_devices);
    if (num_devices > 1) hipSetDevice(DEVICE);

    run(argc,argv);

    return EXIT_SUCCESS;
}

void run(int argc, char** argv)
{
    init(argc, argv);

    /* --------------- pyramid parameters --------------- */
    int borderCols = (pyramid_height)*HALO;
    int smallBlockCol = BLOCK_SIZE-(pyramid_height)*HALO*2;
    int blockCols = cols/smallBlockCol+((cols%smallBlockCol==0)?0:1);
	
    int *gpuWall, *gpuResult[2];
    int size = rows*cols;

    hipMalloc((void**)&gpuResult[0], sizeof(int)*cols);
    hipMalloc((void**)&gpuResult[1], sizeof(int)*cols);
    hipMemcpy(gpuResult[0], data, sizeof(int)*cols, hipMemcpyHostToDevice);
    hipMalloc((void**)&gpuWall, sizeof(int)*(size-cols));
    hipMemcpy(gpuWall, data+cols, sizeof(int)*(size-cols), hipMemcpyHostToDevice);

#ifdef  TIMING
    gettimeofday(&tv_kernel_start, NULL);
#endif

    int final_ret = calc_path(gpuWall, gpuResult, rows, cols, \
	 pyramid_height, blockCols, borderCols);

#ifdef  TIMING
    gettimeofday(&tv_kernel_end, NULL);
    tvsub(&tv_kernel_end, &tv_kernel_start, &tv);
    kernel_time += tv.tv_sec * 1000.0 + (float) tv.tv_usec / 1000.0;
#endif

    hipMemcpy(result, gpuResult[final_ret], sizeof(int)*cols, hipMemcpyDeviceToHost);

    // 读取result.txt文件进行比对
    FILE *file = fopen("result.txt", "r");
    if (file == NULL) {
        printf("Failed\n");
        hipFree(gpuWall);
        hipFree(gpuResult[0]);
        hipFree(gpuResult[1]);
        delete [] data;
        delete [] wall;
        delete [] result;
        return;
    }
    
    int expected_result[cols];
    int i = 0;
    while (fscanf(file, "%d", &expected_result[i]) == 1 && i < cols) {
        i++;
    }
    fclose(file);
    
    // 检查是否读取了足够的元素
    if (i != cols) {
        printf("Failed\n");
        hipFree(gpuWall);
        hipFree(gpuResult[0]);
        hipFree(gpuResult[1]);
        delete [] data;
        delete [] wall;
        delete [] result;
        return;
    }
    
    // 比对结果
    bool match = true;
    for (i = 0; i < cols; i++) {
        if (result[i] != expected_result[i]) {
            match = false;
            break;
        }
    }
    
    
    if (match) {
        printf("Success\n");
    } else {
        printf("Failed\n");
    }

    hipFree(gpuWall);
    hipFree(gpuResult[0]);
    hipFree(gpuResult[1]);

    delete [] data;
    delete [] wall;
    delete [] result;
}
