#include <chrono>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>

#define BLOCK_SIZE 256
#define M_SEED 3608

__global__
void softMax(const int numSlice, const int sliceSize,
             const float* src, float* dest)
{
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= numSlice) return;

  float max_ = src[i * sliceSize];
  for (int j = 0; j < sliceSize; j++) {
    max_ = max(max_, src[i * sliceSize + j]);
  }

  float sum = 0;
  for (int j = 0; j < sliceSize; j++) {
    sum += expf(src[i * sliceSize + j] - max_);
  }

  for (int j = 0; j < sliceSize; j++) {
    dest[i * sliceSize + j] = expf(src[i * sliceSize + j] - max_) / sum;
  }
}

int main(int argc, char* argv[]) {
  if (argc != 3) {
    printf("Usage: %s <number of slices> <slice size>\n", argv[0]);
    return 1;
  }
   
  int numSlice = atoi(argv[1]);
  int sliceSize = atoi(argv[2]);
  int repeat = 1;
  int numElem = numSlice * sliceSize;

  float* input = (float*) aligned_alloc(1024, sizeof(float) * numElem);
  float* output_gpu = (float*) aligned_alloc(1024, sizeof(float) * numElem);

  srand(M_SEED);
  for (int i = 0; i < numSlice; i++)
    for (int j = 0; j < sliceSize; j++)
      input[i*sliceSize+j] = rand() % 13; 

  float *d_input, *d_output;
  hipMalloc((void**)&d_input, sizeof(float) * numElem);
  hipMalloc((void**)&d_output, sizeof(float) * numElem);
  hipMemcpy(d_input, input, sizeof(float) * numElem, hipMemcpyHostToDevice);

  dim3 global_work_size ((numSlice+BLOCK_SIZE-1)/BLOCK_SIZE*BLOCK_SIZE);
  dim3 local_work_size (BLOCK_SIZE);

  hipDeviceSynchronize();

  for (int n = 0; n < repeat; n++) {
    softMax<<<global_work_size, local_work_size>>>(numSlice, sliceSize, d_input, d_output);
  }

  hipDeviceSynchronize();

  hipMemcpy(output_gpu, d_output, sizeof(float) * numElem, hipMemcpyDeviceToHost);

  // ===== 从 result.txt 读取期望结果 =====
  FILE *file = fopen("result.txt", "r");
  if (file == NULL) {
    printf("Failed (cannot open result.txt)\n");
    free(input);
    free(output_gpu);
    hipFree(d_input);
    hipFree(d_output);
    return 1;
  }

  float *expected = (float*) malloc(sizeof(float) * numElem);
  int count = 0;
  while (fscanf(file, "%f", &expected[count]) == 1 && count < numElem) {
    count++;
  }
  fclose(file);

  if (count != numElem) {
    printf("Failed (result.txt does not match expected size)\n");
    free(input);
    free(output_gpu);
    free(expected);
    hipFree(d_input);
    hipFree(d_output);
    return 1;
  }

  // ===== 逐项比对结果 =====
  bool match = true;
  const float eps = 1e-6; // 允许的浮点误差
  for (int i = 0; i < numElem; i++) {
    if (fabs(output_gpu[i] - expected[i]) > eps) {
      match = false;
      break;
    }
  }

  if (match) {
    printf("Success\n");
  } else {
    printf("Failed\n");
  }

  free(input);
  free(output_gpu);
  free(expected);
  hipFree(d_input);
  hipFree(d_output);

  return 0;
}
